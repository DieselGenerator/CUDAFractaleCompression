#include "hip/hip_runtime.h"
#undef NDEBUG

#include <iostream>
#include <vector>
#include <cassert>
#include <cstdint>
#include <inttypes.h>

#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>

#include <opencv2/core.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <opencv2/cudafeatures2d.hpp>
#include <opencv2/cudaarithm.hpp>
#include <opencv2/cudawarping.hpp>
#include <opencv2/cudafilters.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>

#include "ifs_transform_data.hpp"

#include "cuda_call_checker.cuh"
#include "affine_transformer_gpu.cuh"
#include "fractale_compressor_gpu_reduce_ranges.cuh"

__constant__ uint8_t dc_affine_transform_size4[7][16];
__constant__ uint8_t dc_affine_transform_size8[7][64];
__constant__ uint8_t dc_affine_transform_size16[7][256];

void init_fcrr_affine_transformer(int size) {

	uint8_t h_affine_transform_size4[7][16] = {
		//90
		{ 12,  8,  4,  0,
		13,  9,  5,  1,
		14, 10,  6,  2,
		15, 11,  7,  3 },
		//180
		{ 15, 14, 13, 12,
		11, 10,  9,  8,
		7,  6,  5,  4,
		3,  2,  1,  0 },
		//270
		{ 3,  7, 11, 15,
		2,  6, 10, 14,
		1,  5,  9, 13,
		0,  4,  8, 12 },
		//mirror 0
		{ 3,  2,  1,  0,
		7,  6,  5,  4,
		11, 10,  9,  8,
		15, 14, 13, 12 },
		//mirror 90
		{ 15, 11,  7,  3,
		14, 10,  6,  2,
		13,  9,  5,  1,
		12,  8,  4,  0 },
		//mirror 180
		{ 12, 13, 14, 15,
		8,  9, 10, 11,
		4,  5,  6,  7,
		0,  1,  2,  3 },
		//mirror 270
		{ 0,  4,  8, 12,
		1,  5,  9, 13,
		2,  6, 10, 14,
		3,  7, 11, 15 }
	};

	uint8_t h_affine_transform_size8[7][64] = {
		//90
		{ 56, 48, 40, 32, 24, 16,  8,  0,
		57, 49, 41, 33, 25, 17,  9,  1,
		58, 50, 42, 34, 26, 18, 10,  2,
		59, 51, 43, 35, 27, 19, 11,  3,
		60, 52, 44, 36, 28, 20, 12,  4,
		61, 53, 45, 37, 29, 21, 13,  5,
		62, 54, 46, 38, 30, 22, 14,  6,
		63, 55, 47, 39, 31, 23, 15,  7 },
		//180
		{ 63, 62, 61, 60, 59, 58, 57, 56,
		55, 54, 53, 52, 51, 50, 49, 48,
		47, 46, 45, 44, 43, 42, 41, 40,
		39, 38, 37, 36, 35, 34, 33, 32,
		31, 30, 29, 28, 27, 26, 25, 24,
		23, 22, 21, 20, 19, 18, 17, 16,
		15, 14, 13, 12, 11, 10,  9,  8,
		7,  6,  5,  4,  3,  2,  1,  0 },
		//270
		{ 7, 15, 23, 31, 39, 47, 55, 63,
		6, 14, 22, 30, 38, 46, 54, 62,
		5, 13, 21, 29, 37, 45, 53, 61,
		4, 12, 20, 28, 36, 44, 52, 60,
		3, 11, 19, 27, 35, 43, 51, 59,
		2, 10, 18, 26, 34, 42, 50, 58,
		1,  9, 17, 25, 33, 41, 49, 57,
		0,  8, 16, 24, 32, 40, 48, 56 },
		//mirror
		{ 7,  6,  5,  4,  3,  2,  1,  0,
		15, 14, 13, 12, 11, 10,  9,  8,
		23, 22, 21, 20, 19, 18, 17, 16,
		31, 30, 29, 28, 27, 26, 25, 24,
		39, 38, 37, 36, 35, 34, 33, 32,
		47, 46, 45, 44, 43, 42, 41, 40,
		55, 54, 53, 52, 51, 50, 49, 48,
		63, 62, 61, 60, 59, 58, 57, 56 },
		//mirror 90
		{ 63, 55, 47, 39, 31, 23, 15,  7,
		62, 54, 46, 38, 30, 22, 14,  6,
		61, 53, 45, 37, 29, 21, 13,  5,
		60, 52, 44, 36, 28, 20, 12,  4,
		59, 51, 43, 35, 27, 19, 11,  3,
		58, 50, 42, 34, 26, 18, 10,  2,
		57, 49, 41, 33, 25, 17,  9,  1,
		56, 48, 40, 32, 24, 16,  8,  0 },
		//mirror 180
		{ 56, 57, 58, 59, 60, 61, 62, 63,
		48, 49, 50, 51, 52, 53, 54, 55,
		40, 41, 42, 43, 44, 45, 46, 47,
		32, 33, 34, 35, 36, 37, 38, 39,
		24, 25, 26, 27, 28, 29, 30, 31,
		16, 17, 18, 19, 20, 21, 22, 23,
		8,  9, 10, 11, 12, 13, 14, 15,
		0,  1,  2,  3,  4,  5,  6,  7 },
		//mirror 270
		{ 0,  8, 16, 24, 32, 40, 48, 56,
		1,  9, 17, 25, 33, 41, 49, 57,
		2, 10, 18, 26, 34, 42, 50, 58,
		3, 11, 19, 27, 35, 43, 51, 59,
		4, 12, 20, 28, 36, 44, 52, 60,
		5, 13, 21, 29, 37, 45, 53, 61,
		6, 14, 22, 30, 38, 46, 54, 62,
		7, 15, 23, 31, 39, 47, 55, 63 }
	};

	uint8_t h_affine_transform_size16[7][256] = {
		//90
		{ 240,224,208,192,176,160,144,128,112, 96, 80, 64, 48, 32, 16,  0,
		241,225,209,193,177,161,145,129,113, 97, 81, 65, 49, 33, 17,  1,
		242,226,210,194,178,162,146,130,114, 98, 82, 66, 50, 34, 18,  2,
		243,227,211,195,179,163,147,131,115, 99, 83, 67, 51, 35, 19,  3,
		244,228,212,196,180,164,148,132,116,100, 84, 68, 52, 36, 20,  4,
		245,229,213,197,181,165,149,133,117,101, 85, 69, 53, 37, 21,  5,
		246,230,214,198,182,166,150,134,118,102, 86, 70, 54, 38, 22,  6,
		247,231,215,199,183,167,151,135,119,103, 87, 71, 55, 39, 23,  7,
		248,232,216,200,184,168,152,136,120,104, 88, 72, 56, 40, 24,  8,
		249,233,217,201,185,169,153,137,121,105, 89, 73, 57, 41, 25,  9,
		250,234,218,202,186,170,154,138,122,106, 90, 74, 58, 42, 26, 10,
		251,235,219,203,187,171,155,139,123,107, 91, 75, 59, 43, 27, 11,
		252,236,220,204,188,172,156,140,124,108, 92, 76, 60, 44, 28, 12,
		253,237,221,205,189,173,157,141,125,109, 93, 77, 61, 45, 29, 13,
		254,238,222,206,190,174,158,142,126,110, 94, 78, 62, 46, 30, 14,
		255,239,223,207,191,175,159,143,127,111, 95, 79, 63, 47, 31, 15 },
		//180
		{ 255,254,253,252,251,250,249,248,247,246,245,244,243,242,241,240,
		239,238,237,236,235,234,233,232,231,230,229,228,227,226,225,224,
		223,222,221,220,219,218,217,216,215,214,213,212,211,210,209,208,
		207,206,205,204,203,202,201,200,199,198,197,196,195,194,193,192,
		191,190,189,188,187,186,185,184,183,182,181,180,179,178,177,176,
		175,174,173,172,171,170,169,168,167,166,165,164,163,162,161,160,
		159,158,157,156,155,154,153,152,151,150,149,148,147,146,145,144,
		143,142,141,140,139,138,137,136,135,134,133,132,131,130,129,128,
		127,126,125,124,123,122,121,120,119,118,117,116,115,114,113,112,
		111,110,109,108,107,106,105,104,103,102,101,100, 99, 98, 97, 96,
		95, 94, 93, 92, 91, 90, 89, 88, 87, 86, 85, 84, 83, 82, 81, 80,
		79, 78, 77, 76, 75, 74, 73, 72, 71, 70, 69, 68, 67, 66, 65, 64,
		63, 62, 61, 60, 59, 58, 57, 56, 55, 54, 53, 52, 51, 50, 49, 48,
		47, 46, 45, 44, 43, 42, 41, 40, 39, 38, 37, 36, 35, 34, 33, 32,
		31, 30, 29, 28, 27, 26, 25, 24, 23, 22, 21, 20, 19, 18, 17, 16,
		15, 14, 13, 12, 11, 10,  9,  8,  7,  6,  5,  4,  3,  2,  1,  0 },
		//270
		{ 15, 31, 47, 63, 79, 95,111,127,143,159,175,191,207,223,239,255,
		14, 30, 46, 62, 78, 94,110,126,142,158,174,190,206,222,238,254,
		13, 29, 45, 61, 77, 93,109,125,141,157,173,189,205,221,237,253,
		12, 28, 44, 60, 76, 92,108,124,140,156,172,188,204,220,236,252,
		11, 27, 43, 59, 75, 91,107,123,139,155,171,187,203,219,235,251,
		10, 26, 42, 58, 74, 90,106,122,138,154,170,186,202,218,234,250,
		9, 25, 41, 57, 73, 89,105,121,137,153,169,185,201,217,233,249,
		8, 24, 40, 56, 72, 88,104,120,136,152,168,184,200,216,232,248,
		7, 23, 39, 55, 71, 87,103,119,135,151,167,183,199,215,231,247,
		6, 22, 38, 54, 70, 86,102,118,134,150,166,182,198,214,230,246,
		5, 21, 37, 53, 69, 85,101,117,133,149,165,181,197,213,229,245,
		4, 20, 36, 52, 68, 84,100,116,132,148,164,180,196,212,228,244,
		3, 19, 35, 51, 67, 83, 99,115,131,147,163,179,195,211,227,243,
		2, 18, 34, 50, 66, 82, 98,114,130,146,162,178,194,210,226,242,
		1, 17, 33, 49, 65, 81, 97,113,129,145,161,177,193,209,225,241,
		0, 16, 32, 48, 64, 80, 96,112,128,144,160,176,192,208,224,240 },
		//mirror
		{ 15, 14, 13, 12, 11, 10,  9,  8,  7,  6,  5,  4,  3,  2,  1,  0,
		31, 30, 29, 28, 27, 26, 25, 24, 23, 22, 21, 20, 19, 18, 17, 16,
		47, 46, 45, 44, 43, 42, 41, 40, 39, 38, 37, 36, 35, 34, 33, 32,
		63, 62, 61, 60, 59, 58, 57, 56, 55, 54, 53, 52, 51, 50, 49, 48,
		79, 78, 77, 76, 75, 74, 73, 72, 71, 70, 69, 68, 67, 66, 65, 64,
		95, 94, 93, 92, 91, 90, 89, 88, 87, 86, 85, 84, 83, 82, 81, 80,
		111,110,109,108,107,106,105,104,103,102,101,100, 99, 98, 97, 96,
		127,126,125,124,123,122,121,120,119,118,117,116,115,114,113,112,
		143,142,141,140,139,138,137,136,135,134,133,132,131,130,129,128,
		159,158,157,156,155,154,153,152,151,150,149,148,147,146,145,144,
		175,174,173,172,171,170,169,168,167,166,165,164,163,162,161,160,
		191,190,189,188,187,186,185,184,183,182,181,180,179,178,177,176,
		207,206,205,204,203,202,201,200,199,198,197,196,195,194,193,192,
		223,222,221,220,219,218,217,216,215,214,213,212,211,210,209,208,
		239,238,237,236,235,234,233,232,231,230,229,228,227,226,225,224,
		255,254,253,252,251,250,249,248,247,246,245,244,243,242,241,240 },
		//mirror 90
		{ 255,239,223,207,191,175,159,143,127,111, 95, 79, 63, 47, 31, 15,
		254,238,222,206,190,174,158,142,126,110, 94, 78, 62, 46, 30, 14,
		253,237,221,205,189,173,157,141,125,109, 93, 77, 61, 45, 29, 13,
		252,236,220,204,188,172,156,140,124,108, 92, 76, 60, 44, 28, 12,
		251,235,219,203,187,171,155,139,123,107, 91, 75, 59, 43, 27, 11,
		250,234,218,202,186,170,154,138,122,106, 90, 74, 58, 42, 26, 10,
		249,233,217,201,185,169,153,137,121,105, 89, 73, 57, 41, 25,  9,
		248,232,216,200,184,168,152,136,120,104, 88, 72, 56, 40, 24,  8,
		247,231,215,199,183,167,151,135,119,103, 87, 71, 55, 39, 23,  7,
		246,230,214,198,182,166,150,134,118,102, 86, 70, 54, 38, 22,  6,
		245,229,213,197,181,165,149,133,117,101, 85, 69, 53, 37, 21,  5,
		244,228,212,196,180,164,148,132,116,100, 84, 68, 52, 36, 20,  4,
		243,227,211,195,179,163,147,131,115, 99, 83, 67, 51, 35, 19,  3,
		242,226,210,194,178,162,146,130,114, 98, 82, 66, 50, 34, 18,  2,
		241,225,209,193,177,161,145,129,113, 97, 81, 65, 49, 33, 17,  1,
		240,224,208,192,176,160,144,128,112, 96, 80, 64, 48, 32, 16,  0 },
		//mirror 180
		{ 240,241,242,243,244,245,246,247,248,249,250,251,252,253,254,255,
		224,225,226,227,228,229,230,231,232,233,234,235,236,237,238,239,
		208,209,210,211,212,213,214,215,216,217,218,219,220,221,222,223,
		192,193,194,195,196,197,198,199,200,201,202,203,204,205,206,207,
		176,177,178,179,180,181,182,183,184,185,186,187,188,189,190,191,
		160,161,162,163,164,165,166,167,168,169,170,171,172,173,174,175,
		144,145,146,147,148,149,150,151,152,153,154,155,156,157,158,159,
		128,129,130,131,132,133,134,135,136,137,138,139,140,141,142,143,
		112,113,114,115,116,117,118,119,120,121,122,123,124,125,126,127,
		96 ,97, 98, 99,100,101,102,103,104,105,106,107,108,109,110,111,
		80, 81, 82, 83, 84, 85, 86, 87, 88, 89, 90, 91, 92, 93, 94, 95,
		64, 65, 66, 67, 68, 69, 70, 71, 72, 73, 74, 75, 76, 77, 78, 79,
		48, 49, 50, 51, 52, 53, 54, 55, 56, 57, 58, 59, 60, 61, 62, 63,
		32, 33, 34, 35, 36, 37, 38, 39, 40, 41, 42, 43, 44, 45, 46, 47,
		16, 17, 18, 19, 20, 21, 22, 23, 24, 25, 26, 27, 28, 29, 30, 31,
		0,  1,  2,  3,  4,  5,  6,  7,  8,  9, 10, 11, 12, 13, 14, 15 },
		//mirror 270
		{ 0, 16, 32, 48, 64, 80, 96,112,128,144,160,176,192,208,224,240,
		1, 17, 33, 49, 65, 81, 97,113,129,145,161,177,193,209,225,241,
		2, 18, 34, 50, 66, 82, 98,114,130,146,162,178,194,210,226,242,
		3, 19, 35, 51, 67, 83, 99,115,131,147,163,179,195,211,227,243,
		4, 20, 36, 52, 68, 84,100,116,132,148,164,180,196,212,228,244,
		5, 21, 37, 53, 69, 85,101,117,133,149,165,181,197,213,229,245,
		6, 22, 38, 54, 70, 86,102,118,134,150,166,182,198,214,230,246,
		7, 23, 39, 55, 71, 87,103,119,135,151,167,183,199,215,231,247,
		8, 24, 40, 56, 72, 88,104,120,136,152,168,184,200,216,232,248,
		9, 25, 41, 57, 73, 89,105,121,137,153,169,185,201,217,233,249,
		10, 26, 42, 58, 74, 90,106,122,138,154,170,186,202,218,234,250,
		11, 27, 43, 59, 75, 91,107,123,139,155,171,187,203,219,235,251,
		12, 28, 44, 60, 76, 92,108,124,140,156,172,188,204,220,236,252,
		13, 29, 45, 61, 77, 93,109,125,141,157,173,189,205,221,237,253,
		14, 30, 46, 62, 78, 94,110,126,142,158,174,190,206,222,238,254,
		15, 31, 47, 63, 79, 95,111,127,143,159,175,191,207,223,239,255 }
	};

	if (size == 4) {
		CHECK(hipMemcpyToSymbol(HIP_SYMBOL(dc_affine_transform_size4), h_affine_transform_size4, sizeof(uint8_t) * 7 * 16));
		std::cout << "size 4 copyed" << std::endl;
	}
	//cudaMemC
	else if (size == 8) {
		CHECK(hipMemcpyToSymbol(HIP_SYMBOL(dc_affine_transform_size8), h_affine_transform_size8, sizeof(uint8_t) * 7 * 64));
		std::cout << "size 8 copyed" << std::endl;
	}
	else if (size == 16) {
		CHECK(hipMemcpyToSymbol(HIP_SYMBOL(dc_affine_transform_size16), h_affine_transform_size16, sizeof(uint8_t) * 7 * 256));
		std::cout << "size 16 copyed" << std::endl;
	}
	CHECK(hipDeviceSynchronize());
}

/*
input:
	uint8_t* d_orig_img		: ��ʓI�ȉ摜�̌`��
output:
	uint8_t* d_ranges	:�u���b�N�����ꂽ�摜�̌`��
	uint8_t* d_domains	:�u���b�N�����ꂽ�摜�̌`�� �摜�T�C�Y1/2 �u���b�N�T�C�Y�͓���

	�t���N�^�����k�ɕK�v�ȉ摜�z��𐶐�����
	�J�[�l�����ĂԎ��̃u���b�N���Ńh���C���i���k�j�C�����W�̃u���b�N�����肷��
	fc_make_range_n_domain<<<grid, (block_x, block_y)>>>
	��block_x, block_y���u���b�N�̑傫���ɂȂ�
*/
__global__ void fcrr_make_domains_n_ranges(uint8_t* d_orig_img, 
									       uint8_t* d_ranges,
									       uint8_t* d_domains){

	uint32_t rdblock_id = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y;
	uint32_t rdblock_thread_id = blockDim.x * threadIdx.y + threadIdx.x;
	uint32_t rdblock_array_id = rdblock_id + rdblock_thread_id;

	uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
	uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;
	uint32_t normal_array_id = y * (gridDim.x * blockDim.x) + x;

	d_ranges[rdblock_array_id] = d_orig_img[normal_array_id];

	if( (blockIdx.x >= (gridDim.x >> 1) ) || ( blockIdx.y >= (gridDim.y >> 1) ) ){
		return;
	}

	uint32_t half_id = (blockIdx.y * (gridDim.x >> 1) + blockIdx.x) * blockDim.x * blockDim.y;

	uint32_t idx1 = d_orig_img[2 * y * (gridDim.x * blockDim.x) + 2 * x];
	uint32_t idx2 = d_orig_img[2 * y * (gridDim.x * blockDim.x) + 2 * x + 1];
	uint32_t idx3 = d_orig_img[(2 * y + 1) * (gridDim.x * blockDim.x) + 2 * x];
	uint32_t idx4 = d_orig_img[(2 * y + 1) * (gridDim.x * blockDim.x) + 2 * x + 1];

	d_domains[half_id + rdblock_thread_id] = (uint8_t)((idx1 + idx2 + idx3 + idx4) >> 2);
}

/*
	�e�h���C���̑��a�C�ŏ��l�ő�l���v�Z����
	dim3 fc2dblock(dr_blocksize, dr_blocksize, THREADBLOCK_MAX / dr_block_pixel_total);
	dim3 fc2dgrid(dblock_count / fc2dblock.z);
*/
__global__ void fcrr_domain_summimmax(uint8_t* d_domains,
								      uint32_t dblock_count,
									  uint32_t* dblock_sum,
								      uint32_t* dblock_min,
								      uint32_t* dblock_max) 
{
	//sum, min, max��3���ۑ�����
	__shared__ uint32_t domain_summinmax[THREADBLOCK_MAX * 3];
	uint32_t dblock_id = blockIdx.x * blockDim.z + threadIdx.z;
	uint32_t pixel_total = blockDim.x * blockDim.y;
	uint32_t dblock_thread_id = threadIdx.y * blockDim.x + threadIdx.x;
	uint32_t dblock_array_id = dblock_id * pixel_total + dblock_thread_id;

	uint32_t smem_block_id = threadIdx.z;
	uint32_t smem_thread_id = dblock_thread_id;
	uint32_t smem_array_sum_id = smem_block_id * (blockDim.x * blockDim.y) + smem_thread_id;
	uint32_t smem_array_min_id = smem_array_sum_id + THREADBLOCK_MAX;
	uint32_t smem_array_max_id = smem_array_min_id + THREADBLOCK_MAX;

	//if (smem_array_sum_id == THREADBLOCK_MAX) {
	//	printf("asdasfawdfja@opwjgf@paeo");

	//}

	uint8_t pixel = d_domains[dblock_array_id];
	//sum�p
	domain_summinmax[smem_array_sum_id] = pixel;
	//min�p
	domain_summinmax[smem_array_min_id] = pixel;
	//max�p
	domain_summinmax[smem_array_max_id] = pixel;

	__syncthreads();

	for(int32_t i = (blockDim.x * blockDim.y) / 2; i > 0; i >>= 1){
		if(smem_thread_id < i){
			//sum
			domain_summinmax[smem_array_sum_id] += domain_summinmax[smem_array_sum_id + i];
			//min
			if(domain_summinmax[smem_array_min_id] > domain_summinmax[smem_array_min_id + i]){
				domain_summinmax[smem_array_min_id] = domain_summinmax[smem_array_min_id + i];
			}
			//max
			if (domain_summinmax[smem_array_max_id] < domain_summinmax[smem_array_max_id + i]) {
				domain_summinmax[smem_array_max_id] = domain_summinmax[smem_array_max_id + i];
			}
		}
		__syncthreads();
	}

	//�ۑ�
	if (dblock_thread_id == 0) {
		dblock_sum[dblock_id] = domain_summinmax[smem_array_sum_id];
		dblock_min[dblock_id] = domain_summinmax[smem_array_min_id];
		dblock_max[dblock_id] = domain_summinmax[smem_array_max_id];
	};
	__syncthreads();
}

/*
	�e�����W�̑��a�C�ŏ��l�ő�l���v�Z����
	dim3 fc2rblock(dr_blocksize, dr_blocksize, THREADBLOCK_MAX / dr_block_pixel_total);
	dim3 fc2rgrid(rblock_count / fc2rblock.z);
*/
__global__ void fcrr_range_summimmax(uint8_t* d_ranges,
								   uint32_t rblock_count,
								   uint32_t* rblock_sum,
								   uint32_t* rblock_min,
								   uint32_t* rblock_max)
{
	//sum, min, max��3���ۑ�����
	__shared__ uint32_t range_summinmax[THREADBLOCK_MAX * 3];
	uint32_t pixel_total = blockDim.x * blockDim.y;
	uint32_t rblock_id = blockIdx.x * blockDim.z + threadIdx.z;
	uint32_t rblock_thread_id = threadIdx.y * blockDim.x + threadIdx.x;
	uint32_t rblock_array_id = rblock_id * pixel_total + rblock_thread_id;

	uint32_t smem_block_id = threadIdx.z;
	uint32_t smem_thread_id = rblock_thread_id;
	uint32_t smem_array_sum_id = smem_block_id * (blockDim.x * blockDim.y) + smem_thread_id;
	uint32_t smem_array_min_id = smem_array_sum_id + THREADBLOCK_MAX;
	uint32_t smem_array_max_id = smem_array_min_id + THREADBLOCK_MAX;

	uint8_t pixel = d_ranges[rblock_array_id];
	//sum�p
	range_summinmax[smem_array_sum_id] = pixel;
	//min�p
	range_summinmax[smem_array_min_id] = pixel;
	//max�p
	range_summinmax[smem_array_max_id] = pixel;

	__syncthreads();

	for (int32_t i = (blockDim.x * blockDim.y) / 2; i > 0; i >>= 1) {
		if (smem_thread_id < i) {
			//sum
			range_summinmax[smem_array_sum_id] += range_summinmax[smem_array_sum_id + i];
			//min
			if (range_summinmax[smem_array_min_id] > range_summinmax[smem_array_min_id + i]) {
				range_summinmax[smem_array_min_id] = range_summinmax[smem_array_min_id + i];
			}
			//max
			if (range_summinmax[smem_array_max_id] < range_summinmax[smem_array_max_id + i]) {
				range_summinmax[smem_array_max_id] = range_summinmax[smem_array_max_id + i];
			}
		}
		__syncthreads();
	}

	//�ۑ�
	if (rblock_thread_id == 0) {
		//if(rblock_id == 2000){
		//	printf("d_range_value\n");
		//	for (int i = 0; blockDim.x > i; i++) {
		//		printf("%u, %u, %u, %u\n", (uint32_t)d_ranges[rblock_id * pixel_total + (i*4)], (uint32_t)d_ranges[rblock_id * pixel_total + (i * 4 + 1)], (uint32_t)d_ranges[rblock_id * pixel_total + (i * 4 + 2)], (uint32_t)d_ranges[rblock_id * pixel_total + (i * 4 + 3)]);
		//	}
		//	printf("---");
		//	printf("sum : %" PRIu32 "\n", range_summinmax[smem_array_sum_id]);
		//	printf("min : %" PRIu32 "\n", range_summinmax[smem_array_min_id]);
		//	printf("max : %" PRIu32 "\n", range_summinmax[smem_array_max_id]);
		//}
		rblock_sum[rblock_id] = range_summinmax[smem_array_sum_id];
		rblock_min[rblock_id] = range_summinmax[smem_array_min_id];
		rblock_max[rblock_id] = range_summinmax[smem_array_max_id];
	};
	__syncthreads();
}

/*
input:
	uint32_t* d_dblock_sum			:�e�h���C���u���b�N�̑��a
	uint32_t* d_dblock_min			:�e�h���C���u���b�N�̍ŏ��l
	uint32_t* d_dblock_max			:�e�h���C���u���b�N�̍ő�l
	uint32_t* d_rblock_sum			:�e�����W�u���b�N�̑��a
	uint32_t* d_rblock_min			:�e�����W�u���b�N�̍ŏ��l
	uint32_t* d_rblock_max			:�e�����W�u���b�N�̍ő�l
	uint32_t dr_block_pixel_total	:�u���b�N���̉�f��
output:
	double* d_contrast_scaling		:�e�h���C���u���b�N�̊e�����W�u���b�N�ɑ΂���œK�X�P�[�����O
	uint32_t* d_brightness_shift	:�e�h���C���u���b�N�̊e�����W�u���b�N�ɑ΂���œK�P�x�V�t�g

call:
	dim3 fc3block(THREADBLOCK_MAX);
	dim3 fc3grid(dblock_count, rblock_count / THREADBLOCK_MAX);
	fc_calc_scale_n_shift<<<fc3grid, fc3block>>>
	//��̃X���b�h�u���b�N�ŕ����̃����W�u���b�N�̍ŏ��l�C�ő�l���v�Z����
*/
__global__ void fcrr_calc_scale_n_shift(uint32_t* d_dblock_sum,
									  uint32_t* d_dblock_min,
									  uint32_t* d_dblock_max,
									  uint32_t* d_rblock_sum,
									  uint32_t* d_rblock_min,
									  uint32_t* d_rblock_max,
									  uint32_t dr_block_pixel_total,
									  double* d_contrast_scaling,
									  uint32_t* d_brightness_shift)
{
	uint32_t dblock_id = blockIdx.x;
	uint32_t rblock_id = blockIdx.y * blockDim.x + threadIdx.x;
	uint32_t array_id =  blockIdx.x * (gridDim.y * blockDim.x) + rblock_id;

	//�P�x�V�t�g�v�Z
	double shift = (((double)d_rblock_sum[rblock_id] - (double)d_dblock_sum[dblock_id])  / (double)dr_block_pixel_total);

	//if (array_id == 65000000) {
	//	printf("array max :  %" PRIu32 , (gridDim.x * gridDim.y * blockDim.x));
	//	printf("rblock_id  %" PRIu32 "dblock_id %" PRIu32 "\n", rblock_id, dblock_id);
	//	printf("rblock_id_sum[r]  %" PRIu32 "dblock_id_sum[d] %" PRIu32 "\n", d_rblock_sum[rblock_id], d_dblock_sum[dblock_id]);
	//	printf("rblock_id_sum[d]  %" PRIu32 "dblock_id_sum[d] %" PRIu32 "\n", d_rblock_sum[dblock_id], d_dblock_sum[dblock_id]);
	//	printf("contrast %lf, shift : %" PRIu32, d_contrast_scaling[array_id], d_brightness_shift[array_id]);
	//}


	if (shift < 0) {
		shift = 0;
	}
	else if (shift > 255) {
		shift = 255;
	}

	d_brightness_shift[array_id] = (uint32_t)shift;
	//�R���g���X�g�X�P�[�����O
	double d = (double)(d_dblock_max[dblock_id] - d_dblock_min[dblock_id]);
	double r = (double)(d_rblock_max[rblock_id] - d_rblock_min[rblock_id]);

	double raw_scaling = r / d;

	//uint8_t scaling_save = 0;
	for (double j = 0.0625; j <= 1; j += 0.0625) {
		if ((j - 0.0625) < raw_scaling && raw_scaling < j) {
			d_contrast_scaling[array_id] = j;
			break;
		}
		d_contrast_scaling[array_id] = 1;
	}



	//d_contrast_scaling[array_id] = raw_scaling;


	/*
		TODO ��{�I��4bit���ɏk������K�v���L��ׁC
		�X�P�[�����O�̏��͈��k���ĕێ������K�v������
	*/

	//double min;
	//double max;
	//uint32_t scaling;
	//for (min = -0.03125, max = 0.03125, scaling = 0; scaling < 16; min += 0.0625, max += 0.0625, scaling++) {
	//	if (min < raw_scaling && raw_scaling < max){
	//		d_brightness_shift[array_id] = scaling;
	//		return;
	//	}
	//}
	////0.9625�ȏ�͑S��15�ɁE�E�E�H
	//d_brightness_shift[array_id] = 0xF;
}

/*
	dim3 fc4block(dr_blocksize, dr_blocksize, THREADBLOCK_MAX / dr_block_pixel_total);
	dim3 fc4grid(dblock_count , rblock_count / fc4block.z);
	fc_transform_n_calc_mse<<<fc4grid, fc4block>>>
*/
__global__ void fcrr_transform_n_calc_mse(uint8_t* d_domains,
										uint8_t* d_ranges,
										double* d_contrast_scaling,
										uint32_t* d_brightness_shift,
										uint32_t* d_mse,
										bool is_inner,
										uint32_t periphery,
										uint32_t rblock_cols,
										uint32_t rblock_rows)
{
	__shared__ uint32_t mse_all[THREADBLOCK_MAX];

	uint32_t drblock_thread_id = threadIdx.y * blockDim.x + threadIdx.x;
	uint32_t drblock_pixel_total = blockDim.x * blockDim.y;
	uint32_t dblock_id = blockIdx.x;
	uint32_t dblock_count = gridDim.x;
	uint32_t rblock_id = blockIdx.y * blockDim.z + threadIdx.z;
	uint32_t rblock_count = gridDim.y * blockDim.z;

	/*���̃X���b�h���O��������S�����Ă��邩�ǂ����̃t���O*/
	bool is_this_thread_outer = (rblock_id < rblock_cols * periphery/*�㕔*/ ||
								rblock_id >= rblock_count - rblock_cols * periphery/*����*/ ||
								(rblock_id % rblock_cols) < periphery /*����*/ ||
								(rblock_id % rblock_cols) >= rblock_cols - periphery)/*�E��*/;

	if (is_inner == is_this_thread_outer) {
		return;
	}

	/*
	if (is_inner) {
		if (is_this_thread_outer){
			return;
		}
	}
	else {
		if (!is_this_thread_outer) {
			return;
		}
	}
	*/

	uint32_t array_id = dblock_id * rblock_count + rblock_id;

	uint32_t smem_array_id = threadIdx.z * drblock_pixel_total + drblock_thread_id;
	uint32_t smem_block_id = threadIdx.z * drblock_pixel_total;
	uint32_t smem_thread_id = drblock_thread_id;

	//���̃����W�Ƀh���C���ɓK������scaling, shift
	uint32_t shift = d_brightness_shift[array_id];
	double scale = d_contrast_scaling[array_id];

	//�����W
	uint8_t rpixel = d_ranges[rblock_id * drblock_pixel_total + drblock_thread_id];
	int32_t rpixel2 = (int32_t)rpixel * (int32_t)rpixel;
	double f_dpixel;
	for (int32_t rotate = 0; rotate < 8; rotate++) {
		if (rotate == 0){
			f_dpixel = (scale * (double)d_domains[dblock_id * (drblock_pixel_total) + drblock_thread_id]) + (double)shift;
		}
		else {
			if (blockDim.x == 4) {
				f_dpixel = (scale * (double)d_domains[dblock_id * (drblock_pixel_total) + (uint32_t)dc_affine_transform_size4[rotate - 1][drblock_thread_id]]) + (double)shift;
			}
			else if (blockDim.x == 8) {
				f_dpixel = (scale * (double)d_domains[dblock_id * (drblock_pixel_total) + (uint32_t)dc_affine_transform_size8[rotate - 1][drblock_thread_id]]) + (double)shift;
			}
			else if (blockDim.x == 16) {
				f_dpixel = (scale * (double)d_domains[dblock_id * (drblock_pixel_total) + (uint32_t)dc_affine_transform_size16[rotate - 1][drblock_thread_id]]) + (double)shift;
			}
		}
		if (f_dpixel > 255) {
			f_dpixel = 255;
		} else if (f_dpixel < 0) {
			f_dpixel = 0;
		}
		
		int32_t f_dpixel2 = (int32_t)f_dpixel * (int32_t)f_dpixel;
		int32_t diff = rpixel2 - f_dpixel2;
		diff = diff < 0 ? -diff : diff;
		uint32_t diff_abs = diff;
		mse_all[smem_array_id] = diff_abs;

		__syncthreads();

		for (int32_t j = drblock_pixel_total / 2; j > 0; j >>= 1) {
			if(smem_thread_id < j){
				mse_all[smem_array_id] += mse_all[smem_array_id + j];
			}
			__syncthreads();
		}
		if(smem_thread_id == 0){
			d_mse[rotate * dblock_count * rblock_count + dblock_id * rblock_count + rblock_id] = mse_all[smem_array_id];
		}
		__syncthreads();
	}
}

/*
	�e�����W�̓��덷���ŏ��l�ł���h���C��(�h���܂�)��index�����_�N�V�����ŋ��߁C�e�W����ێ�����
	�e�X���b�h���Ɂ@�ŏ��ƂȂ�mse���v�Z���Ă���
	dim3 fc5block(THREADBLOCK_MAX);
	dim3 fc5grid(rblock_count/ THREADBLOCK_MAX);
*/
__global__ void fcrr_save_min_mse(uint32_t dblock_cols,
								uint32_t dblock_rows,
								uint32_t blocksize,
								uint32_t* d_mse, 
								double* d_cotrast_scaling, 
								uint32_t* d_brightness_shift, 
								compress_data_part_reduce_ranges_gpu* d_compress_data_part_gpu,
								bool is_inner,
								uint32_t periphery,
								uint32_t rblock_cols,
								uint32_t rblock_rows) 
{
	uint32_t rblock_id = blockDim.x * blockIdx.x + threadIdx.x;
	uint32_t rblock_count = gridDim.x * blockDim.x;
	uint32_t dblock_count = dblock_cols * dblock_rows;
	
	uint32_t best_mse = UINT32_MAX;

	/*���̃X���b�h���O��������S�����Ă��邩�ǂ����̃t���O*/
	bool is_this_thread_outer = (rblock_id < rblock_cols * periphery/*�㕔*/ ||
								rblock_id >= rblock_count - rblock_cols * periphery/*����*/ ||
								(rblock_id % rblock_cols) < periphery /*����*/ ||
								(rblock_id % rblock_cols) >= rblock_cols - periphery)/*�E��*/;

	if (is_inner == is_this_thread_outer) {
		return;
	}

	for (uint32_t dblock_y = 0; dblock_y < dblock_rows; dblock_y++) {
		for (uint32_t dblock_x = 0; dblock_x < dblock_cols; dblock_x++) {
			for (uint8_t rotate = 0; rotate < 7; rotate++) {
				uint32_t dblock_id = dblock_cols * dblock_y + dblock_x;
				uint32_t array_id = dblock_id * rblock_count + rblock_id;

				/*
					���}���u
				*/
				//bool medic = !((dblock_id * 4 == rblock_id) & (dblock_id * 4 == rblock_id + 1) & (dblock_id * 4 == rblock_id + 2) & (dblock_id * 4 == rblock_id + 3));
				bool medic = !(((rblock_id / blocksize) == dblock_id));

				if (d_mse[rotate * dblock_count * rblock_count + array_id] < best_mse & medic) {
					best_mse = d_mse[rotate * dblock_count * rblock_count + array_id];
					d_compress_data_part_gpu[rblock_id].rblock_id = rblock_id;
					d_compress_data_part_gpu[rblock_id].dblock_id = dblock_id;
					d_compress_data_part_gpu[rblock_id].rotate = rotate;
					d_compress_data_part_gpu[rblock_id].scale = d_cotrast_scaling[array_id];
					d_compress_data_part_gpu[rblock_id].shift = d_brightness_shift[array_id];
				}
				//if (rotate == 4 && dblock_x == 0 && dblock_y == 0 && blockIdx.x == 0 && threadIdx.x == 999) {
				//	printf("reached rotate == 3 value : %" PRIu32"\n best value : %" PRIu32"\n", d_mse[rotate * dblock_count * rblock_count + array_id], best_mse);
				//}
			}
		}
	}

}

/*
	CPU��
	�S�̂�4x4�C�u���b�N�T�C�Y��2x2�̎��C��f�l�̕��т�1���ȉ��̔z��̕��тɂ���
	{ 0, 1, 2, 3,
	  4, 5, 6, 6,
	  8, 9,10,11,
	 12,13,14,14} -> {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15}
*/
void fcrr_img2array(cv::Mat img, uint8_t* img_array) {
	assert(img.isContinuous());
	img.convertTo(img, CV_8UC1);


	for (uint32_t y = 0; y < img.rows; y++) {
		for (uint32_t x = 0; x < img.cols; x++) {
			img_array[y*img.rows + x] = img.at<uint8_t>(y, x);
		}
	}
}

/*
input:
	cv::Mat img			: ��ʓI�ȉ摜�̌`��
	uint32_t blocksize	:�u���b�N�̑傫��
	bool is_inner		:�����̃u���b�N����������J�[�l���ł��邩
	periphery			:�O���������𖳎�or�̂ݎ��s���邩�D
return;
	std::vector<ifs_transformer*> : �o�͕�����

	is_inner��true�̏ꍇ�C�O��periphery���l�������������ĉ��Z���s���i�X���b�h�͋N�����邪�C�����͋�j
	is_inner��false�̏ꍇ�C�O�������̂ݎ��s����

	�t���N�^�����k�̈��k���s����A��GPU�J�[�l�����Ă�
	img�𕄍�������
*/
std::vector<ifs_transformer*> launch_reduce_ranges_compress_kernel(cv::Mat img, uint32_t blocksize, bool is_inner, uint32_t periphery)
{	
	/*
		0.�O�����
	*/

	//�h���C���E�����W�u���b�N�̂P�ӂ̒���
	uint32_t dr_blocksize = blocksize;
	//�u���b�N1���܂މ�f��
	uint32_t dr_block_pixel_total = dr_blocksize * dr_blocksize;
	//�����W�u���b�N�̕ӓ�����̐�
	uint32_t rblock_cols = img.cols / dr_blocksize;
	uint32_t rblock_rows = img.rows / dr_blocksize;
	uint32_t rblock_count = rblock_cols * rblock_rows;
	//�h���C���u���b�N�̕ӕӂ�̐�
	uint32_t dblock_cols = rblock_cols >> 1;
	uint32_t dblock_rows = rblock_rows >> 1;
	uint32_t dblock_count = dblock_cols * dblock_rows;

	assert(blocksize == 4 || blocksize == 8 || blocksize == 16);
	assert((img.cols % blocksize) == 0);
	assert((img.rows % blocksize) == 0);
	assert(dblock_count % (THREADBLOCK_MAX / blocksize) == 0);
	assert(rblock_count % (THREADBLOCK_MAX / blocksize) == 0);
	assert(((rblock_count * dblock_count) % THREADBLOCK_MAX) == 0);
	assert(rblock_count < (65535 * THREADBLOCK_MAX));

	std::cout << "domain block count : " << dblock_count << std::endl;
	std::cout << "range block count : " << rblock_count << std::endl;

	/*
		1.�u���b�N�ϊ��E�k���ϊ�
	*/

	//�u���b�N�ϊ��E�k���ϊ��ׂ̈̋N���X���b�h���ݒ�
	dim3 fc1block(dr_blocksize, dr_blocksize);
	dim3 fc1grid(rblock_cols, rblock_rows);
	std::cout << "reduction grid :" << " x = " << fc1grid.x << " y = " << fc1grid.y << std::endl;
	uint32_t orig_arraysize = img.total() * img.channels();
	uint8_t* h_orig_img = new uint8_t[orig_arraysize];
	uint8_t* d_orig_img;
	uint8_t* d_ranges;
	uint8_t* d_domains;
	
	CHECK(hipMalloc((void**)&d_orig_img, sizeof(uint8_t) * orig_arraysize));
	CHECK(hipMalloc((void**)&d_ranges, sizeof(uint8_t) * orig_arraysize));
	CHECK(hipMalloc((void**)&d_domains, sizeof(uint8_t) * orig_arraysize >> 2));

	fcrr_img2array(img, h_orig_img);

	CHECK(hipMemcpy(d_orig_img, h_orig_img, sizeof(uint8_t) * orig_arraysize, hipMemcpyHostToDevice));
	fcrr_make_domains_n_ranges<<<fc1grid, fc1block>>>(d_orig_img, d_ranges, d_domains);
	CHECK(hipDeviceSynchronize());

	//uint8_t* h_ranges = new uint8_t[orig_arraysize];
	//uint8_t* h_domains = new uint8_t[orig_arraysize >> 2];
	//CHECK(hipMemcpy(h_ranges, d_ranges, sizeof(uint8_t) * orig_arraysize, hipMemcpyDeviceToHost));
	//CHECK(hipMemcpy(h_domains, d_domains, sizeof(uint8_t) * orig_arraysize >> 2, hipMemcpyDeviceToHost));
	//show_img2(h_ranges, img.cols, img.rows, dr_blocksize);
	//show_img2(h_domains, img.cols >> 1, img.rows >> 1, dr_blocksize);

	/*
		2.�h���C���E�����W�̑��a�E�ŏ��l�ő�l�v�Z
	*/

	//�P�̃X���b�h�u���b�N�ŕ����̃h���C������������
	dim3 fc2dblock(dr_blocksize, dr_blocksize, THREADBLOCK_MAX / dr_block_pixel_total);
	dim3 fc2dgrid(dblock_count / fc2dblock.z);

	uint32_t* d_dblock_sum;
	uint32_t* d_dblock_min;
	uint32_t* d_dblock_max;
	CHECK(hipMalloc((void**)&d_dblock_sum, sizeof(uint32_t) * dblock_count));
	CHECK(hipMalloc((void**)&d_dblock_min, sizeof(uint32_t) * dblock_count));
	CHECK(hipMalloc((void**)&d_dblock_max, sizeof(uint32_t) * dblock_count));
	fcrr_domain_summimmax<<<fc2dgrid, fc2dblock>>>(d_domains, dblock_count, d_dblock_sum, d_dblock_min, d_dblock_max);

	//�P�̃X���b�h�u���b�N�ŕ����̃����W����������
	dim3 fc2rblock(dr_blocksize, dr_blocksize, THREADBLOCK_MAX / dr_block_pixel_total);
	dim3 fc2rgrid(rblock_count / fc2rblock.z);
	uint32_t* d_rblock_sum;
	uint32_t* d_rblock_min;
	uint32_t* d_rblock_max;
	CHECK(hipMalloc((void**)&d_rblock_sum, sizeof(uint32_t) * rblock_count));
	CHECK(hipMalloc((void**)&d_rblock_min, sizeof(uint32_t) * rblock_count));
	CHECK(hipMalloc((void**)&d_rblock_max, sizeof(uint32_t) * rblock_count));
	fcrr_range_summimmax<<<fc2rgrid, fc2rblock>>>(d_ranges, rblock_count, d_rblock_sum, d_rblock_min, d_rblock_max);

	CHECK(hipDeviceSynchronize());

	/*
		3.�R���g���X�g�X�P�[�����O�E�P�x�V�t�g�v�Z
	*/

	dim3 fc3block(THREADBLOCK_MAX);
	dim3 fc3grid(dblock_count, rblock_count / THREADBLOCK_MAX);

	double* d_contrast_scaling;
	uint32_t* d_brightness_shift;

	CHECK(hipMalloc((void**)&d_contrast_scaling, sizeof(double) * dblock_count * rblock_count));
	CHECK(hipMalloc((void**)&d_brightness_shift, sizeof(uint32_t) * dblock_count * rblock_count));
	//CHECK(hipMalloc((void**)&d_adjust_domains_for_ranges, sizeof(uint32_t) * dblock_count * rblock_count * dr_block_pixel_total));

	//std::cout << "fc3grid : " << fc3grid.operator uint3 << "fc3block : " << fc3grid << std::endl;

	fcrr_calc_scale_n_shift<<<fc3grid, fc3block>>>(d_dblock_sum,
												 d_dblock_min,
												 d_dblock_max,
												 d_rblock_sum,
									     		 d_rblock_min,
												 d_rblock_max,
												 dr_block_pixel_total,
											     d_contrast_scaling,
											     d_brightness_shift);

	CHECK(hipDeviceSynchronize());

	/*
		4.�R���g���X�g�X�P�[�����O�E�P�x�V�t�g�K�p�E��]�E�����ϊ��E���v�Z�EMSE�v�Z�i���_�N�V�����j
	*/
	//�u���b�N�T�C�Y�ɉ������R���X�^���g������������������
	init_fcrr_affine_transformer(dr_blocksize);
	CHECK(hipDeviceSynchronize());
	dim3 fc4block(dr_blocksize, dr_blocksize, THREADBLOCK_MAX / dr_block_pixel_total);
	dim3 fc4grid(dblock_count , rblock_count / fc4block.z);
	//�e�����W�̊e�h���C���̊e��]�ϊ����MSE��ۑ����Ă���
	uint32_t* d_mse;
	CHECK(hipMalloc((void**)&d_mse, sizeof(uint32_t) * rblock_count * dblock_count * 8));

	fcrr_transform_n_calc_mse<<<fc4grid, fc4block>>>(d_domains,
												     d_ranges,
												     d_contrast_scaling,
												     d_brightness_shift,
												     d_mse,
													 is_inner,
													 periphery,
													 rblock_cols,
													 rblock_rows);

	CHECK(hipDeviceSynchronize());
	//std::cout << sizeof(uint32_t) * rblock_count * dblock_count * 8 << "byte" << std::endl;

	//uint32_t* h_mse = new uint32_t[rblock_count * dblock_count * 8];
	//CHECK(hipMemcpy(h_mse, d_mse, sizeof(uint32_t) * rblock_count * dblock_count * 8, hipMemcpyDeviceToHost));

	//std::cout << "nukiuti : " << h_mse[231] << std::endl;

	//delete[] h_mse;

	/*
		5.�e�����W���ŏ�MSE�Eindex�v�Z(���_�N�V��������)
	*/

	dim3 fc5block(THREADBLOCK_MAX);
	dim3 fc5grid(rblock_count/ THREADBLOCK_MAX);

	//�����W�̐��������k�f�[�^��ۑ�����
	compress_data_part_reduce_ranges_gpu* h_compress_data = new compress_data_part_reduce_ranges_gpu[rblock_count];
	compress_data_part_reduce_ranges_gpu* d_compress_data;
	CHECK(hipMalloc((void**)&d_compress_data, sizeof(compress_data_part_reduce_ranges_gpu) * rblock_count));

	fcrr_save_min_mse<<<fc5grid, fc5block>>>(dblock_cols,
											 dblock_rows,
											 blocksize, d_mse, 
											 d_contrast_scaling,
											 d_brightness_shift, 
											 d_compress_data,
											 is_inner,
											 periphery,
											 rblock_cols,
											 rblock_rows);
	CHECK(hipDeviceSynchronize());
	CHECK(hipMemcpy(h_compress_data, d_compress_data, sizeof(compress_data_part_reduce_ranges_gpu) * rblock_count, hipMemcpyDeviceToHost));

	/*
		6.HOST���̃f�[�^�����ʂ̌`���ɉ��H
	*/

	CHECK(hipDeviceSynchronize());

	std::vector<ifs_transformer*> ifs_data;

	for (int32_t i = 0; i < rblock_count; i++) {
		/*���̃X���b�h���O��������S�����Ă��邩�ǂ����̃t���O*/
		bool is_this_thread_outer = (i < rblock_cols * periphery/*�㕔*/ ||
			i >= rblock_count - rblock_cols * periphery/*����*/ ||
			(i % rblock_cols) < periphery /*����*/ ||
			(i % rblock_cols) >= rblock_cols - periphery)/*�E��*/;

		if (is_inner == is_this_thread_outer) {
			continue;
		}

		//std::cout << h_compress_data[i].rotate << std::endl;
		ifs_transformer* c = new ifs_transformer();
		c->error = std::numeric_limits<double>::max();
		c->rblock_x = (h_compress_data[i].rblock_id % rblock_rows)*blocksize;
		c->rblock_y = (h_compress_data[i].rblock_id / rblock_rows)*blocksize;
		c->dblock_x = (h_compress_data[i].dblock_id % dblock_rows)*blocksize;
		c->dblock_y = (h_compress_data[i].dblock_id / dblock_rows)*blocksize;
		c->affine = h_compress_data[i].rotate;

		//std::cout << "h_comp:  " << h_compress_data[i].scale << std::endl;

		uint8_t scaling_save = 0;
		for (double j = 0.0625; j < 1; j += 0.0625) {
			//if ((j - 0.0625) <= h_compress_data[i].scale && h_compress_data[i].scale < j) {
			//	scaling_save = ((j) * 16);
			//	break;
			//}
			if (j == h_compress_data[i].scale) {
				break;
			}
			scaling_save++;
		}
		c->scaling = scaling_save;
		c->shift = h_compress_data[i].shift;
		c->blocksize = blocksize;
		ifs_data.push_back(c);
	}

	//std::cout << "just test : " << (int32_t)h_affine_transform_size4_1d[0] << std::endl;

	//delete[] h_ranges;
	//delete[] h_domains;

	/*
		L.�㏈��
	*/

	delete[] h_orig_img;
	delete[] h_compress_data;

	CHECK(hipFree(d_orig_img));
	CHECK(hipFree(d_ranges));
	CHECK(hipFree(d_domains));

	CHECK(hipFree(d_rblock_sum));
	CHECK(hipFree(d_rblock_min));
	CHECK(hipFree(d_rblock_max));
	
	CHECK(hipFree(d_dblock_sum));
	CHECK(hipFree(d_dblock_min));
	CHECK(hipFree(d_dblock_max));

	CHECK(hipFree(d_contrast_scaling));
	CHECK(hipFree(d_brightness_shift));

	CHECK(hipFree(d_mse));

	CHECK(hipDeviceSynchronize());

	return ifs_data;
}

/*

	fc_make_range_n_domain<<<resize_grid, resize_block>>>(d_orig_img, d, d)


	dim3 resize_block(16, 16);
	dim3 resize_grid(((img.cols >> 1) + resize_block.x - 1) / resize_block.x, ((img.rows >> 1) + resize_block.y - 1) / resize_block.y);
	std::cout << "reduction grid :" << " x = " << resize_grid.x << " y = " << resize_grid.y << std::endl;
	
	//���摜��z��Ƃ��ĕ\���ꍇ�̔z��T�C�Y(�P��byte)�G
	uint32_t orig_img_array_size = img.total() * img.channels();
	uint8_t* h_orig_img = new uint8_t[orig_img_array_size];
	uint8_t* d_orig_img;
	CHECK(hipMalloc((void**)&d_orig_img, sizeof(uint8_t) * orig_img_array_size));
	//�k����摜��z��Ƃ��ĕ\���ꍇ�̔z��T�C�Y(�P��byte);
	uint32_t stage1_size = img.total() * img.channels() >> 2;
	uint8_t* h_stage1 = new uint8_t[stage1_size];
	uint8_t* d_stage1;
	CHECK(hipMalloc((void**)&d_stage1, sizeof(uint8_t) * stage1_size));

	//(img, h_orig_img);
	//show_img(h_orig_img, img.cols, img.rows);
	img2blockarray(img, h_orig_img, 32);
	show_img2(h_orig_img, img.rows, img.cols, 32);
	CHECK(hipMemcpy(d_orig_img, h_orig_img, sizeof(uint8_t) * orig_img_array_size, hipMemcpyHostToDevice));
	fc_resize2<<<resize_grid, resize_block>>>(d_orig_img, d_stage1, img.cols);
	//fc_resize<<<resize_grid, resize_block>>>(d_orig_img, d_stage1, img.cols);
	CHECK(hipMemcpy(h_stage1, d_stage1, sizeof(uint8_t) * stage1_size, hipMemcpyDeviceToHost));
	show_img2(h_stage1 ,(img.rows >> 1), (img.cols >> 1), 16);
	
	/*
		2.�P�x�V�t�g, �R���g���X�g�X�P�[�����O
	*/

	//dim3 resize_block(32, 32);
	//dim3 resize_grid(1, 1);
/*

	//test_kernel<<<1, 1>>>();
	CHECK(hipDeviceSynchronize());

	delete[] h_orig_img;
	delete[] h_stage1;
*/