
#include <hip/hip_runtime.h>
//#undef NDEBUG
//
//#include <iostream>
//#include <cassert>
//
//#include <inttypes.h>
//
//#include <cuda_runtime.h>
//#include <device_launch_parameters.h>
//#include <device_functions.h>
//
//#include <opencv2/core.hpp>
//#include <opencv2/core/cuda.hpp>
//#include <opencv2/cudaimgproc.hpp>
//#include <opencv2/cudafeatures2d.hpp>
//#include <opencv2/cudaarithm.hpp>
//#include <opencv2/cudawarping.hpp>
//#include <opencv2/cudafilters.hpp>
//#include <opencv2/imgproc.hpp>
//#include <opencv2/imgcodecs.hpp>
//#include <opencv2/highgui.hpp>
//
//#include "cuda_call_checker.cuh"
//#include "affine_transformer_gpu.cuh"
//#include "fractale_compressor_gpu_rgb.cuh"
//
///*
//	全体が4x4，ブロックサイズが2x2の時
//	{ 0, 1, 2, 3,     { 0, 1, 4, 5,
//	  4, 5, 6, 6,       2, 3, 6, 7,
//	  8, 9,10,11,       8, 9,12,13,
//	 12,13,14,14} ->   10,11,14,15}
//	 と並び替える，ブロックサイズはカーネル呼び出しで変更する
//	 fc_arrangement<<<(grid_x, grid_y), (block_x, block_y, n)>>>(i, o, size);
//	 各ブロック毎に
//*/
////いらん？
////__global__ void fc_arrangement(uint8_t* d_original_img, uint8_t* d_arrangement_img, uint32_t block_size){
////	extern uint8_t sm[];
////	
////	//ブロックの数
////	uint32_t blocks_num = gridDim.x;
////	//ブロックの大きさ
////	uint32_t block_total = blockDim.x * blockDim.y;
////
////
////
////	//ブロックの配列の先頭index
////	uint32_t dst_block_index = (blockIdx.y * blocks_num + blockIdx.x) * block_total;
////	//スレッドのindex
////	uint32_t dst_thread_index = threadIdx.y * blockDim.y + threadIdx.x;
////
////	//各ブロックの画素値を一時的に保持する
////	sm[dst_block_index + dst_thread_index];
////
////	__syncthreads();
////}
//
///*
//	一般的な画素の並びの配列に保存された画像を縦横1/2倍に縮小する
//	4点の平均値を算出するだけの方式
//*/
//__global__ void fc_resize(uint8_t* d_original_img,
//						  uint8_t* d_resize_img,
//						  uint32_t original_width) 
//{
//	uint32_t xx = threadIdx.x + blockIdx.x * blockDim.x;
//	uint32_t yy = threadIdx.y + blockIdx.y * blockDim.y;
//	uint32_t ix = xx << 1;
//	uint32_t iy = yy << 1;
//
//	uint32_t idx1 = d_original_img[(iy * original_width) + ix];
//	uint32_t idx2 = d_original_img[(iy * original_width) + ix + 1];
//	uint32_t idx3 = d_original_img[((iy + 1) * original_width) + ix];
//	uint32_t idx4 = d_original_img[((iy + 1) * original_width) + ix + 1];
//
//	d_resize_img[yy * (original_width >> 1)+ xx] = (uint8_t)((idx1 + idx2 + idx3 + idx4) >> 2);
//}
//
///*
//input:
//	uint8_t* d_orig_img		: 一般的な画像の形式
//output:
//	uint8_t* d_ranges	:ブロック化された画像の形式
//	uint8_t* d_domains	:ブロック化された画像の形式 画像サイズ1/2 ブロックサイズは同じ
//
//	フラクタル圧縮に必要な画像配列を生成する
//	カーネルを呼ぶ時のブロック数でドメイン（圧縮），レンジのブロックを決定する
//	fc_make_range_n_domain<<<grid, (block_x, block_y)>>>
//	のblock_x, block_yがブロックの大きさになる
//*/
//__global__ void fc_make_domains_n_ranges(uint8_t* d_orig_img, 
//									     uint8_t* d_ranges,
//									     uint8_t* d_domains){
//
//	uint32_t rdblock_id = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y;
//	uint32_t rdblock_thread_id = blockDim.x * threadIdx.y + threadIdx.x;
//	uint32_t rdblock_array_id = rdblock_id + rdblock_thread_id;
//
//	uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
//	uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;
//	uint32_t normal_array_id = y * (gridDim.x * blockDim.x) + x;
//
//	d_ranges[rdblock_array_id] = d_orig_img[normal_array_id];
//
//	if( (blockIdx.x >= (gridDim.x >> 1) ) || ( blockIdx.y >= (gridDim.y >> 1) ) ){
//		return;
//	}
//
//	uint32_t half_id = (blockIdx.y * (gridDim.x >> 1) + blockIdx.x) * blockDim.x * blockDim.y;
//
//	uint32_t idx1 = d_orig_img[2 * y * (gridDim.x * blockDim.x) + 2 * x];
//	uint32_t idx2 = d_orig_img[2 * y * (gridDim.x * blockDim.x) + 2 * x + 1];
//	uint32_t idx3 = d_orig_img[(2 * y + 1) * (gridDim.x * blockDim.x) + 2 * x];
//	uint32_t idx4 = d_orig_img[(2 * y + 1) * (gridDim.x * blockDim.x) + 2 * x + 1];
//
//	d_domains[half_id + rdblock_thread_id] = (uint8_t)((idx1 + idx2 + idx3 + idx4) >> 2);
//}
//
///*
//	各ドメインの総和，最小値最大値を計算する
//*/
//__global__ void fc_domain_summimmax(uint8_t* d_domains,
//								    uint32_t dblock_count,
//									uint32_t* dblock_sum,
//								    uint32_t* dblock_min,
//								    uint32_t* dblock_max) 
//{
//	//sum, min, maxの3種を保存する
//	__shared__ uint32_t domain_summinmax[THREADBLOCK_MAX * 3];
//	uint32_t dblock_id = blockIdx.x * blockDim.z + threadIdx.z;
//	uint32_t dblock_thread_id = threadIdx.y * blockDim.x + threadIdx.x;
//	uint32_t dblock_array_id = dblock_id + dblock_thread_id;
//
//	uint32_t smem_block_id = threadIdx.z;
//	uint32_t smem_thread_id = dblock_thread_id;
//	uint32_t smem_array_sum_id = smem_block_id * (blockDim.x * blockDim.y) + smem_thread_id;
//	uint32_t smem_array_min_id = smem_array_sum_id + THREADBLOCK_MAX;
//	uint32_t smem_array_max_id = smem_array_min_id + THREADBLOCK_MAX;
//
//	if (smem_array_sum_id == THREADBLOCK_MAX) {
//		printf("asdasfawdfja@opwjgf@paeo");
//
//	}
//
//	uint8_t pixel = d_domains[dblock_array_id];
//	//sum用
//	domain_summinmax[smem_array_sum_id] = pixel;
//	//min用
//	domain_summinmax[smem_array_min_id] = pixel;
//	//max用
//	domain_summinmax[smem_array_max_id] = pixel;
//
//	__syncthreads();
//
//	for(int32_t i = (blockDim.x * blockDim.y) / 2; i > 0; i >>= 1){
//		if(smem_thread_id < i){
//			//sum
//			domain_summinmax[smem_array_sum_id] += domain_summinmax[smem_array_sum_id + i];
//			//min
//			if(domain_summinmax[smem_array_min_id] > domain_summinmax[smem_array_min_id + i]){
//				domain_summinmax[smem_array_min_id] = domain_summinmax[smem_array_min_id + i];
//			}
//			//max
//			if (domain_summinmax[smem_array_max_id] < domain_summinmax[smem_array_max_id + i]) {
//				domain_summinmax[smem_array_max_id] = domain_summinmax[smem_array_max_id + i];
//			}
//		}
//		__syncthreads();
//	}
//
//	//保存
//	if (dblock_thread_id == 0) {
//		dblock_sum[dblock_id] = domain_summinmax[smem_array_sum_id];
//		dblock_min[dblock_id] = domain_summinmax[smem_array_min_id];
//		dblock_max[dblock_id] = domain_summinmax[smem_array_max_id];
//	};
//}
//
///*
//	各レンジの総和，最小値最大値を計算する
//
//
//*/
//__global__ void fc_range_summimmax(uint8_t* d_ranges,
//								   uint32_t rblock_count,
//								   uint32_t* rblock_sum,
//								   uint32_t* rblock_min,
//								   uint32_t* rblock_max)
//{
//	//sum, min, maxの3種を保存する
//	__shared__ uint32_t range_summinmax[THREADBLOCK_MAX * 3];
//	uint32_t rblock_id = blockIdx.x * blockDim.z + threadIdx.z;
//	uint32_t rblock_thread_id = threadIdx.y * blockDim.x + threadIdx.x;
//	uint32_t rblock_array_id = rblock_id + rblock_thread_id;
//
//	uint32_t smem_block_id = threadIdx.z;
//	uint32_t smem_thread_id = rblock_thread_id;
//	uint32_t smem_array_sum_id = smem_block_id * (blockDim.x * blockDim.y) + smem_thread_id;
//	uint32_t smem_array_min_id = smem_array_sum_id + THREADBLOCK_MAX;
//	uint32_t smem_array_max_id = smem_array_min_id + THREADBLOCK_MAX;
//
//	uint8_t pixel = d_ranges[rblock_array_id];
//	//sum用
//	range_summinmax[smem_array_sum_id] = pixel;
//	//min用
//	range_summinmax[smem_array_min_id] = pixel;
//	//max用
//	range_summinmax[smem_array_max_id] = pixel;
//
//	__syncthreads();
//
//	for (int32_t i = (blockDim.x * blockDim.y) / 2; i > 0; i >>= 1) {
//		if (smem_thread_id < i) {
//			//sum
//			range_summinmax[smem_array_sum_id] += range_summinmax[smem_array_sum_id + i];
//			//min
//			if (range_summinmax[smem_array_min_id] > range_summinmax[smem_array_min_id + i]) {
//				range_summinmax[smem_array_min_id] = range_summinmax[smem_array_min_id + i];
//			}
//			//max
//			if (range_summinmax[smem_array_max_id] < range_summinmax[smem_array_max_id + i]) {
//				range_summinmax[smem_array_max_id] = range_summinmax[smem_array_max_id + i];
//			}
//		}
//		__syncthreads();
//	}
//
//	//保存
//	if (rblock_thread_id == 0) {
//		//if(rblock_id == 0){
//		//	printf("sum : %" PRIu32 "\n", range_summinmax[smem_array_sum_id]);
//		//	printf("min : %" PRIu32 "\n", range_summinmax[smem_array_min_id]);
//		//	printf("max : %" PRIu32 "\n", range_summinmax[smem_array_max_id]);
//		//}
//		rblock_sum[rblock_id] = range_summinmax[smem_array_sum_id];
//		rblock_min[rblock_id] = range_summinmax[smem_array_min_id];
//		rblock_max[rblock_id] = range_summinmax[smem_array_max_id];
//	};
//}
//
///*
//input:
//	uint32_t* d_dblock_sum			:各ドメインブロックの総和
//	uint32_t* d_dblock_min			:各ドメインブロックの最小値
//	uint32_t* d_dblock_max			:各ドメインブロックの最大値
//	uint32_t* d_rblock_sum			:各レンジブロックの総和
//	uint32_t* d_rblock_min			:各レンジブロックの最小値
//	uint32_t* d_rblock_max			:各レンジブロックの最大値
//	uint32_t dr_block_pixel_total	:ブロック内の画素数
//output:
//	double* d_contrast_scaling		:各ドメインブロックの各レンジブロックに対する最適スケーリング
//	uint32_t* d_brightness_shift	:各ドメインブロックの各レンジブロックに対する最適輝度シフト
//
//call:
//	dim3 fc3block(THREADBLOCK_MAX);
//	dim3 fc3grid(dblock_count, rblock_count / THREADBLOCK_MAX);
//	fc_calc_scale_n_shift<<<fc3grid, fc3block>>>
//	//一つのスレッドブロックで複数のレンジブロックの最小値，最大値を計算する
//*/
//__global__ void fc_calc_scale_n_shift(uint32_t* d_dblock_sum,
//									  uint32_t* d_dblock_min,
//									  uint32_t* d_dblock_max,
//									  uint32_t* d_rblock_sum,
//									  uint32_t* d_rblock_min,
//									  uint32_t* d_rblock_max,
//									  uint32_t dr_block_pixel_total,
//									  double* d_contrast_scaling,
//									  uint32_t* d_brightness_shift)
//{
//	uint32_t dblock_id = blockIdx.x;
//	uint32_t rblock_id = blockIdx.y * blockDim.x + threadIdx.x;
//	uint32_t array_id =  blockIdx.x * (gridDim.y * blockDim.x) + rblock_id;
//
//	//輝度シフト計算
//	int32_t shift = ((int32_t)d_dblock_sum[dblock_id] - (int32_t)d_rblock_sum[rblock_id])  / dr_block_pixel_total;
//	d_brightness_shift[array_id] = shift;
//	//コントラストスケーリング
//	double d = (double)(d_dblock_max[dblock_id] - d_dblock_min[dblock_id]);
//	double r = (double)(d_rblock_max[rblock_id] - d_rblock_min[rblock_id]);
//
//	double raw_scaling = r / d;
//	d_contrast_scaling[array_id] = raw_scaling;
//
//	/*
//		TODO 基本的に4bit内に縮小する必要が有る為，
//		スケーリングの情報は圧縮して保持される必要がある
//	*/
//
//	//double min;
//	//double max;
//	//uint32_t scaling;
//	//for (min = -0.03125, max = 0.03125, scaling = 0; scaling < 16; min += 0.0625, max += 0.0625, scaling++) {
//	//	if (min < raw_scaling && raw_scaling < max){
//	//		d_brightness_shift[array_id] = scaling;
//	//		return;
//	//	}
//	//}
//	////0.9625以上は全部15に・・・？
//	//d_brightness_shift[array_id] = 0xF;
//}
//
///*
//	dim3 fc4block(dr_blocksize, dr_blocksize, THREADBLOCK_MAX / dr_block_pixel_total);
//	dim3 fc4grid(dblock_count , rblock_count / fc4block.z);
//	fc_transform_n_calc_mse<<<fc4grid, fc4block>>>
//*/
//__global__ void fc_transform_n_calc_mse(uint8_t* d_domains,
//										uint8_t* d_ranges,
//										double* d_contrast_scaling,
//										uint32_t* d_brightness_shift,
//										uint32_t* mse)
//{
//	__shared__ uint32_t mse_all[THREADBLOCK_MAX];
//
//	uint32_t drblock_thread_id = threadIdx.y * blockDim.x + threadIdx.x;
//	uint32_t drblock_pixel_total = blockDim.x * blockDim.y;
//	uint32_t dblock_id = blockIdx.x;
//	uint32_t dblock_count = gridDim.x;
//	uint32_t rblock_id = blockIdx.y * blockDim.z + threadIdx.z;
//	uint32_t rblock_count = gridDim.y * blockDim.z;
//
//	uint32_t array_id = dblock_id * rblock_count + rblock_id;
//
//	uint32_t smem_array_id = threadIdx.z * drblock_pixel_total + drblock_thread_id;
//	uint32_t smem_block_id = threadIdx.z * drblock_pixel_total;
//	uint32_t smem_thread_id = drblock_thread_id;
//
//	//このレンジにドメインに適応するscaling, shift
//	uint32_t shift = d_brightness_shift[array_id];
//	double scale = d_contrast_scaling[array_id];
//
//	double f_d_p = scale * (double)d_domains[dblock_id + drblock_thread_id] + (double)shift;
//
//	if(f_d_p < 0){
//		f_d_p = -f_d_p;
//	}
//	//uint8_tにキャストした時点で最大値は255に固定される
//	uint8_t fixed_dpixel = (uint8_t)f_d_p;
//	int32_t fixed_dpixel2 = (int32_t)fixed_dpixel * (int32_t)fixed_dpixel;
//	//オリジナルレンジ
//	uint8_t rpixel = d_ranges[rblock_id + drblock_thread_id];
//
//	for(int32_t rotate = 0; rotate < 8; rotate++){
//		int32_t fixed_dpixel2 = (int32_t)fixed_dpixel * (int32_t)fixed_dpixel;
//		int32_t rpixel2 = (int32_t)rpixel * (int32_t)rpixel;
//		int32_t diff = fixed_dpixel2 - rpixel2;
//		diff = diff < 0 ? -diff : diff;
//		uint32_t diff_abs = diff;
//		mse_all[smem_array_id] = diff_abs;
//
//		for (int32_t j = drblock_pixel_total / 2; j > 0; j >>= 1) {
//			if(smem_thread_id < j){
//				mse_all[smem_array_id] += mse_all[smem_array_id + j];
//			}
//			__syncthreads();
//		}
//		if(smem_thread_id == 0){
//			mse[rotate * dblock_count * rblock_count + dblock_id * rblock_count + rblock_id] = mse_all[smem_array_id];
//		}
//		if (rotate < 7) {
//			if (blockDim.x == 4) {
//				rpixel = d_ranges[rblock_id + dc_affine_transform_size4[rotate][drblock_thread_id]];
//			}
//			else if (blockDim.x == 8) {
//				rpixel = d_ranges[rblock_id + dc_affine_transform_size8[rotate][drblock_thread_id]];
//			}
//			else if (blockDim.x == 16) {
//				rpixel = d_ranges[rblock_id + dc_affine_transform_size16[rotate][drblock_thread_id]];
//			}
//		}
//		__syncthreads();
//	}
//}
//
///*
//	各レンジの二乗誤差が最小値であるドメイン(派生含む)のindexをリダクションで求め，各係数を保持する
//*/
//__global__ void fc_save_min_mse(uint32_t* d_mse, 
//								double* d_cotrast_scaling, 
//								uint32_t* d_brightness_shift, 
//								compress_data_part_rgb_gpu* d_compress_data_part_gpu) 
//{
//	
//}
//
///*
//	//
//	画像を縦横1/2倍に縮小する
//	4点の平均値を算出するだけの方式
//	カーネルでブロックサイズを合わせる
//	元ブロックサイズを
//	元カネールの
//	fc_resize2<<<(grid_x, grid_y), (block_x, block_y, n)>>>(i, o, size);
//*/
//__global__ void fc_resize2(uint8_t* d_original_img, uint8_t* d_resize_img, uint32_t original_width) {
//	//blockDim.x, blockDim.yはリサイズ後のブロックサイズ
//	//blockの数自体は変換前後で一定
//	uint32_t blocks_num = gridDim.x;//original_width / blockDim.x;
//
//	//resize後ブロックの大きさ
//	uint32_t resize_block_total = blockDim.x * blockDim.y;
//	//resize後ブロックの配列の先頭index
//	uint32_t resize_block_index = (blockIdx.y * blocks_num + blockIdx.x) * resize_block_total;
//	//resize後ブロック内スレッドのindex
//	uint32_t resize_thread_index = threadIdx.y * blockDim.y + threadIdx.x;
//
//	//元ブロックの大きさ
//	uint32_t orig_block_total = resize_block_total << 2;
//	//元ブロックの大きさの配列の先頭index
//	uint32_t orig_block_index = (blockIdx.y * blocks_num + blockIdx.x) * orig_block_total;
//	//resize後ブロック内スレッドのindex1
//	uint32_t orig_thread_index1 = (threadIdx.y << 1) * (blockDim.y << 1) + (threadIdx.x << 1);
//	//resize後ブロック内スレッドのindex2
//	uint32_t orig_thread_index2 = (threadIdx.y << 1) * (blockDim.y << 1) + (threadIdx.x << 1) + 1;
//	//resize後ブロック内スレッドのindex3
//	uint32_t orig_thread_index3 = ((threadIdx.y << 1) + 1) * (blockDim.y << 1) + (threadIdx.x << 1);
//	//resize後ブロック内スレッドのindex4
//	uint32_t orig_thread_index4 = ((threadIdx.y << 1) + 1) * (blockDim.y << 1) + (threadIdx.x << 1) + 1;
//
//	//resize後ブロック内スレッド
//	uint32_t idx1 = d_original_img[orig_block_index + orig_thread_index1];
//	uint32_t idx2 = d_original_img[orig_block_index + orig_thread_index2];
//	uint32_t idx3 = d_original_img[orig_block_index + orig_thread_index3];
//	uint32_t idx4 = d_original_img[orig_block_index + orig_thread_index4];
//
//	d_resize_img[resize_block_index + resize_thread_index] = (uint8_t)((idx1 + idx2 + idx3 + idx4) >> 2);
//}
//
///*
//	全体が4x4，ブロックサイズが2x2の時
//	{ 0, 1, 2, 3,
//	  4, 5, 6, 6,
//	  8, 9,10,11,
//	 12,13,14,14}
//	のような画素値の並びのグレースケールの配列画像をテスト表示する
//	全画素を代入していくので速度は遅い，テスト用
//*/
//void show_img(uint8_t* img_array, uint32_t width, uint32_t height){
//	cv::Mat mat(width, height, CV_8U);
//	for (uint32_t y = 0; y < height; y++) {
//		for (uint32_t x = 0; x < width; x++) {
//			mat.at<uint8_t>(y, x) = img_array[y*width + x];
//		}
//	}
//	cv::namedWindow("show_img", cv::WINDOW_AUTOSIZE);
//	cv::imshow("show_img", mat);
//	cv::waitKey(0);
//	cv::destroyAllWindows();
//}
//
///*
//	全体が4x4，ブロックサイズが2x2の時
//	{ 0, 1, 4, 5,
//	  2, 3, 6, 7,
//	  8, 9,12,13,
//	 10,11,14,15}
//	のような画素値の並びのグレースケールの配列画像をテスト表示する
//	全画素を代入していくので速度は遅い，テスト用
//	block_size == block_height == block_width
//*/
//void show_img2(uint8_t* img_array, uint32_t width, uint32_t height, uint32_t block_size) {
//	cv::Mat mat(width, height, CV_8U);
//
//	//横方向のレンジの数
//	uint32_t range_x_n = width / block_size;
//	//縦方向のレンジの数
//	uint32_t range_y_n = height / block_size;
//
//	uint32_t index = 0;
//	for(uint32_t y = 0; y < range_y_n * block_size; y += block_size){
//		for (uint32_t x = 0; x < range_x_n * block_size; x += block_size) {
//			//各ブロック
//			for (uint32_t block_rows = 0; block_rows < block_size; block_rows++) {
//				for (uint32_t block_cols = 0; block_cols < block_size; block_cols++) {
//					mat.at<uint8_t>(y + block_rows, x + block_cols) = img_array[index];
//					index++;
//				}
//			}
//		}
//	}
//
//	cv::namedWindow("show_img", cv::WINDOW_AUTOSIZE);
//	cv::imshow("show_img", mat);
//	cv::waitKey(0);
//	cv::destroyAllWindows();
//}
//
//
///*
//	CPU版
//	全体が4x4，ブロックサイズが2x2の時，画素値の並びを1次以下の配列の並びにする
//	{ 0, 1, 2, 3,
//	  4, 5, 6, 6,
//	  8, 9,10,11,
//	 12,13,14,14} -> {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15}
//*/
//void img2array(cv::Mat img, uint8_t* img_array) {
//	assert(img.isContinuous());
//	img.convertTo(img, CV_8UC1);
//
//
//	for (uint32_t y = 0; y < img.rows; y++) {
//		for (uint32_t x = 0; x < img.cols; x++) {
//			img_array[y*img.rows + x] = img.at<uint8_t>(y, x);
//		}
//	}
//}
//
///*
//	CPU版
//	全体が4x4，ブロックサイズが2x2の時，画素値の並びを1次以下の配列の並びにする
//	{ 0, 1, 2, 3,
//	  4, 5, 6, 7,
//	  8, 9,10,11,
//	 12,13,14,15} -> {0, 1, 4, 5, 2, 3, 6, 7, 8, 9, 12, 13, 10, 11, 14, 15}
//*/
//void img2blockarray(cv::Mat img, uint8_t* img_array, uint32_t block_size){
//	assert(img.isContinuous());
//	assert((img.cols % block_size) == 0);
//	assert((img.rows % block_size) == 0);
//	img.convertTo(img, CV_8UC1);
//
//	//横方向のレンジの数
//	uint32_t range_x_n = img.cols / block_size;
//	//縦方向のレンジの数
//	uint32_t range_y_n = img.rows / block_size;
//	std::cout << "tatal : "<<img.total() << std::endl;
//	uint32_t index = 0;
//	for (uint32_t y = 0; y < range_y_n * block_size; y += block_size) {
//		for (uint32_t x = 0; x < range_x_n * block_size; x += block_size) {
//			//各ブロック
//			
//			for (uint32_t block_rows = 0; block_rows < block_size; block_rows++) {
//				for (uint32_t block_cols = 0; block_cols < block_size; block_cols++) {
//					
//					//if(index % 1000 == 0)std::cout << index << std::endl;
//					img_array[index] = img.at<uint8_t>(y + block_rows, x + block_cols);
//					index++;
//				}
//			}
//		}
//	}
//
//}
//
//void launch_rgb_compress_kernel(cv::Mat img, uint32_t blocksize)
//{	
//	/*
//		0.前提条件
//	*/
//
//	//ドメイン・レンジブロックの１辺の長さ
//	uint32_t dr_blocksize = blocksize;
//	//ブロック1つが含む画素数
//	uint32_t dr_block_pixel_total = dr_blocksize * dr_blocksize;
//	//レンジブロックの辺当たりの数
//	uint32_t rblock_cols = img.cols / dr_blocksize;
//	uint32_t rblock_rows = img.rows / dr_blocksize;
//	uint32_t rblock_count = rblock_cols * rblock_rows;
//	//ドメインブロックの辺辺りの数
//	uint32_t dblock_cols = rblock_cols >> 1;
//	uint32_t dblock_rows = rblock_rows >> 1;
//	uint32_t dblock_count = dblock_cols * dblock_rows;
//
//	assert(blocksize == 4 || blocksize == 8 || blocksize == 16);
//	assert((img.cols % blocksize) == 0);
//	assert((img.rows % blocksize) == 0);
//	assert(dblock_count % (THREADBLOCK_MAX / blocksize) == 0);
//	assert(rblock_count % (THREADBLOCK_MAX / blocksize) == 0);
//	assert(((rblock_count * dblock_count) % THREADBLOCK_MAX) == 0);
//	assert(rblock_count < (65535 * THREADBLOCK_MAX));
//
//	std::cout << "domain block count : " << dblock_count << std::endl;
//	std::cout << "range block count : " << rblock_count << std::endl;
//
//	/*
//		1.ブロック変換・縮小変換
//	*/
//	//ブロック変換・縮小変換の為の起動スレッド数設定
//	dim3 fc1block(dr_blocksize, dr_blocksize);
//	dim3 fc1grid(rblock_cols, rblock_rows);
//	std::cout << "reduction grid :" << " x = " << fc1grid.x << " y = " << fc1grid.y << std::endl;
//	uint32_t orig_arraysize = img.total() * img.channels();
//	uint8_t* h_orig_img = new uint8_t[orig_arraysize];
//	uint8_t* d_orig_img;
//	uint8_t* d_ranges;
//	uint8_t* d_domains;
//	
//	CHECK(cudaMalloc((void**)&d_orig_img, sizeof(uint8_t) * orig_arraysize));
//	CHECK(cudaMalloc((void**)&d_ranges, sizeof(uint8_t) * orig_arraysize));
//	CHECK(cudaMalloc((void**)&d_domains, sizeof(uint8_t) * orig_arraysize >> 2));
//
//	img2array(img, h_orig_img);
//
//	CHECK(cudaMemcpy(d_orig_img, h_orig_img, sizeof(uint8_t) * orig_arraysize, cudaMemcpyHostToDevice));
//	fc_make_domains_n_ranges<<<fc1grid, fc1block>>>(d_orig_img, d_ranges, d_domains);
//	CHECK(cudaDeviceSynchronize());
//	//uint8_t* h_ranges = new uint8_t[orig_arraysize];
//	//uint8_t* h_domains = new uint8_t[orig_arraysize >> 2];
//	//CHECK(cudaMemcpy(h_ranges, d_ranges, sizeof(uint8_t) * orig_arraysize, cudaMemcpyDeviceToHost));
//	//CHECK(cudaMemcpy(h_domains, d_domains, sizeof(uint8_t) * orig_arraysize >> 2, cudaMemcpyDeviceToHost));
//	//show_img2(h_ranges, img.cols, img.rows, dr_blocksize);
//	//show_img2(h_domains, img.cols >> 1, img.rows >> 1, dr_blocksize);
//
//	/*
//		2.ドメイン・レンジの総和・最小値最大値計算
//	*/
//	//１つのスレッドブロックで複数のドメインを処理する
//	dim3 fc2dblock(dr_blocksize, dr_blocksize, THREADBLOCK_MAX / dr_block_pixel_total);
//	dim3 fc2dgrid(dblock_count / fc2dblock.z);
//
//	uint32_t* d_dblock_sum;
//	uint32_t* d_dblock_min;
//	uint32_t* d_dblock_max;
//	CHECK(cudaMalloc((void**)&d_dblock_sum, sizeof(uint32_t) * dblock_count));
//	CHECK(cudaMalloc((void**)&d_dblock_min, sizeof(uint32_t) * dblock_count));
//	CHECK(cudaMalloc((void**)&d_dblock_max, sizeof(uint32_t) * dblock_count));
//	fc_domain_summimmax<<<fc2dgrid, fc2dblock>>>(d_domains, dblock_count, d_dblock_sum, d_dblock_min, d_dblock_max);
//
//	//１つのスレッドブロックで複数のレンジを処理する
//	dim3 fc2rblock(dr_blocksize, dr_blocksize, THREADBLOCK_MAX / dr_block_pixel_total);
//	dim3 fc2rgrid(rblock_count / fc2rblock.z);
//	uint32_t* d_rblock_sum;
//	uint32_t* d_rblock_min;
//	uint32_t* d_rblock_max;
//	CHECK(cudaMalloc((void**)&d_rblock_sum, sizeof(uint32_t) * rblock_count));
//	CHECK(cudaMalloc((void**)&d_rblock_min, sizeof(uint32_t) * rblock_count));
//	CHECK(cudaMalloc((void**)&d_rblock_max, sizeof(uint32_t) * rblock_count));
//	fc_range_summimmax<<<fc2rgrid, fc2rblock>>>(d_ranges, rblock_count, d_rblock_sum, d_rblock_min, d_rblock_max);
//
//	CHECK(cudaDeviceSynchronize());
//
//	/*
//		3.コントラストスケーリング・輝度シフト計算
//	*/
//
//	dim3 fc3block(THREADBLOCK_MAX);
//	dim3 fc3grid(dblock_count, rblock_count / THREADBLOCK_MAX);
//
//	double* d_contrast_scaling;
//	uint32_t* d_brightness_shift;
//
//	CHECK(cudaMalloc((void**)&d_contrast_scaling, sizeof(double) * dblock_count * rblock_count));
//	CHECK(cudaMalloc((void**)&d_brightness_shift, sizeof(uint32_t) * dblock_count * rblock_count));
//	//CHECK(cudaMalloc((void**)&d_adjust_domains_for_ranges, sizeof(uint32_t) * dblock_count * rblock_count * dr_block_pixel_total));
//
//	fc_calc_scale_n_shift<<<fc3grid, fc3block>>>(d_dblock_sum,
//												 d_dblock_min,
//												 d_dblock_max,
//												 d_rblock_sum,
//									     		 d_rblock_min,
//												 d_rblock_max,
//												 dr_block_pixel_total,
//											     d_contrast_scaling,
//											     d_brightness_shift);
//
//	CHECK(cudaDeviceSynchronize());
//
//	/*
//		4.コントラストスケーリング・輝度シフト適用・回転・鏡像変換・二乗計算・MSE計算（リダクション）
//	*/
//
//	//
//	dim3 fc4block(dr_blocksize, dr_blocksize, THREADBLOCK_MAX / dr_block_pixel_total);
//	dim3 fc4grid(dblock_count , rblock_count / fc4block.z);
//	//各レンジの各ドメインの各回転変換後のMSEを保存しておく
//	uint32_t* d_mse;
//	CHECK(cudaMalloc((void**)&d_mse, sizeof(uint32_t) * rblock_count * dblock_count * 8));
//
//	fc_transform_n_calc_mse<<<fc4grid, fc4block>>>(d_domains,
//												   d_ranges,
//												   d_contrast_scaling,
//												   d_brightness_shift,
//												   d_mse);
//	CHECK(cudaDeviceSynchronize());
//	//std::cout << sizeof(uint32_t) * rblock_count * dblock_count * 8 << "byte" << std::endl;
//
//	//uint32_t* h_mse = new uint32_t[rblock_count * dblock_count * 8];
//	//CHECK(cudaMemcpy(h_mse, d_mse, sizeof(uint32_t) * rblock_count * dblock_count * 8, cudaMemcpyDeviceToHost));
//
//	//std::cout << "nukiuti : " << h_mse[231] << std::endl;
//
//	//delete[] h_mse;
//
//	/*
//		5.各レンジ毎最小MSE・index計算(リダクション)
//	*/
//	//ブロックサイズに応じたコンスタントメモリを初期化する
//	init_affine_transformer(dr_blocksize);
//	dim3 fc5block(THREADBLOCK_MAX);
//	dim3 fc5grid(dblock_count * 8, rblock_count/ THREADBLOCK_MAX);
//
//	//レンジの数だけ圧縮データを保存する
//	compress_data_part_rgb_gpu* h_compress_data = new compress_data_part_rgb_gpu[rblock_count];
//	compress_data_part_rgb_gpu* d_compress_data;
//	CHECK(cudaMalloc((void**)&d_compress_data, sizeof(compress_data_part_rgb_gpu) * rblock_count));
//
//	fc_save_min_mse<<<fc5grid, fc5block>>>(d_mse, d_contrast_scaling, d_brightness_shift, d_compress_data);
//	
//	CHECK(cudaMemcpy(h_compress_data, d_compress_data, sizeof(compress_data_part_rgb_gpu) * rblock_count, cudaMemcpyDeviceToHost));
//
//	//delete[] h_ranges;
//	//delete[] h_domains;
//
//	/*
//		L.後処理
//	*/
//
//	delete[] h_orig_img;
//	delete[] h_compress_data;
//
//	CHECK(cudaFree(d_orig_img));
//	CHECK(cudaFree(d_ranges));
//	CHECK(cudaFree(d_domains));
//
//	CHECK(cudaFree(d_rblock_sum));
//	CHECK(cudaFree(d_rblock_min));
//	CHECK(cudaFree(d_rblock_max));
//	
//	CHECK(cudaFree(d_dblock_sum));
//	CHECK(cudaFree(d_dblock_min));
//	CHECK(cudaFree(d_dblock_max));
//
//	CHECK(cudaFree(d_contrast_scaling));
//	CHECK(cudaFree(d_brightness_shift));
//
//	CHECK(cudaFree(d_mse));
//
//	CHECK(cudaDeviceSynchronize());
//}
