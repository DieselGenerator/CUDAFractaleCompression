
#include <hip/hip_runtime.h>
//#undef NDEBUG
//
//#include <iostream>
//#include <cassert>
//
//#include <inttypes.h>
//
//#include <cuda_runtime.h>
//#include <device_launch_parameters.h>
//#include <device_functions.h>
//
//#include <opencv2/core.hpp>
//#include <opencv2/core/cuda.hpp>
//#include <opencv2/cudaimgproc.hpp>
//#include <opencv2/cudafeatures2d.hpp>
//#include <opencv2/cudaarithm.hpp>
//#include <opencv2/cudawarping.hpp>
//#include <opencv2/cudafilters.hpp>
//#include <opencv2/imgproc.hpp>
//#include <opencv2/imgcodecs.hpp>
//#include <opencv2/highgui.hpp>
//
//#include "cuda_call_checker.cuh"
//#include "affine_transformer_gpu.cuh"
//#include "fractale_compressor_gpu_rgb.cuh"
//
///*
//	�S�̂�4x4�C�u���b�N�T�C�Y��2x2�̎�
//	{ 0, 1, 2, 3,     { 0, 1, 4, 5,
//	  4, 5, 6, 6,       2, 3, 6, 7,
//	  8, 9,10,11,       8, 9,12,13,
//	 12,13,14,14} ->   10,11,14,15}
//	 �ƕ��ёւ���C�u���b�N�T�C�Y�̓J�[�l���Ăяo���ŕύX����
//	 fc_arrangement<<<(grid_x, grid_y), (block_x, block_y, n)>>>(i, o, size);
//	 �e�u���b�N����
//*/
////�����H
////__global__ void fc_arrangement(uint8_t* d_original_img, uint8_t* d_arrangement_img, uint32_t block_size){
////	extern uint8_t sm[];
////	
////	//�u���b�N�̐�
////	uint32_t blocks_num = gridDim.x;
////	//�u���b�N�̑傫��
////	uint32_t block_total = blockDim.x * blockDim.y;
////
////
////
////	//�u���b�N�̔z��̐擪index
////	uint32_t dst_block_index = (blockIdx.y * blocks_num + blockIdx.x) * block_total;
////	//�X���b�h��index
////	uint32_t dst_thread_index = threadIdx.y * blockDim.y + threadIdx.x;
////
////	//�e�u���b�N�̉�f�l���ꎞ�I�ɕێ�����
////	sm[dst_block_index + dst_thread_index];
////
////	__syncthreads();
////}
//
///*
//	��ʓI�ȉ�f�̕��т̔z��ɕۑ����ꂽ�摜���c��1/2�{�ɏk������
//	4�_�̕��ϒl���Z�o���邾���̕���
//*/
//__global__ void fc_resize(uint8_t* d_original_img,
//						  uint8_t* d_resize_img,
//						  uint32_t original_width) 
//{
//	uint32_t xx = threadIdx.x + blockIdx.x * blockDim.x;
//	uint32_t yy = threadIdx.y + blockIdx.y * blockDim.y;
//	uint32_t ix = xx << 1;
//	uint32_t iy = yy << 1;
//
//	uint32_t idx1 = d_original_img[(iy * original_width) + ix];
//	uint32_t idx2 = d_original_img[(iy * original_width) + ix + 1];
//	uint32_t idx3 = d_original_img[((iy + 1) * original_width) + ix];
//	uint32_t idx4 = d_original_img[((iy + 1) * original_width) + ix + 1];
//
//	d_resize_img[yy * (original_width >> 1)+ xx] = (uint8_t)((idx1 + idx2 + idx3 + idx4) >> 2);
//}
//
///*
//input:
//	uint8_t* d_orig_img		: ��ʓI�ȉ摜�̌`��
//output:
//	uint8_t* d_ranges	:�u���b�N�����ꂽ�摜�̌`��
//	uint8_t* d_domains	:�u���b�N�����ꂽ�摜�̌`�� �摜�T�C�Y1/2 �u���b�N�T�C�Y�͓���
//
//	�t���N�^�����k�ɕK�v�ȉ摜�z��𐶐�����
//	�J�[�l�����ĂԎ��̃u���b�N���Ńh���C���i���k�j�C�����W�̃u���b�N�����肷��
//	fc_make_range_n_domain<<<grid, (block_x, block_y)>>>
//	��block_x, block_y���u���b�N�̑傫���ɂȂ�
//*/
//__global__ void fc_make_domains_n_ranges(uint8_t* d_orig_img, 
//									     uint8_t* d_ranges,
//									     uint8_t* d_domains){
//
//	uint32_t rdblock_id = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y;
//	uint32_t rdblock_thread_id = blockDim.x * threadIdx.y + threadIdx.x;
//	uint32_t rdblock_array_id = rdblock_id + rdblock_thread_id;
//
//	uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
//	uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;
//	uint32_t normal_array_id = y * (gridDim.x * blockDim.x) + x;
//
//	d_ranges[rdblock_array_id] = d_orig_img[normal_array_id];
//
//	if( (blockIdx.x >= (gridDim.x >> 1) ) || ( blockIdx.y >= (gridDim.y >> 1) ) ){
//		return;
//	}
//
//	uint32_t half_id = (blockIdx.y * (gridDim.x >> 1) + blockIdx.x) * blockDim.x * blockDim.y;
//
//	uint32_t idx1 = d_orig_img[2 * y * (gridDim.x * blockDim.x) + 2 * x];
//	uint32_t idx2 = d_orig_img[2 * y * (gridDim.x * blockDim.x) + 2 * x + 1];
//	uint32_t idx3 = d_orig_img[(2 * y + 1) * (gridDim.x * blockDim.x) + 2 * x];
//	uint32_t idx4 = d_orig_img[(2 * y + 1) * (gridDim.x * blockDim.x) + 2 * x + 1];
//
//	d_domains[half_id + rdblock_thread_id] = (uint8_t)((idx1 + idx2 + idx3 + idx4) >> 2);
//}
//
///*
//	�e�h���C���̑��a�C�ŏ��l�ő�l���v�Z����
//*/
//__global__ void fc_domain_summimmax(uint8_t* d_domains,
//								    uint32_t dblock_count,
//									uint32_t* dblock_sum,
//								    uint32_t* dblock_min,
//								    uint32_t* dblock_max) 
//{
//	//sum, min, max��3���ۑ�����
//	__shared__ uint32_t domain_summinmax[THREADBLOCK_MAX * 3];
//	uint32_t dblock_id = blockIdx.x * blockDim.z + threadIdx.z;
//	uint32_t dblock_thread_id = threadIdx.y * blockDim.x + threadIdx.x;
//	uint32_t dblock_array_id = dblock_id + dblock_thread_id;
//
//	uint32_t smem_block_id = threadIdx.z;
//	uint32_t smem_thread_id = dblock_thread_id;
//	uint32_t smem_array_sum_id = smem_block_id * (blockDim.x * blockDim.y) + smem_thread_id;
//	uint32_t smem_array_min_id = smem_array_sum_id + THREADBLOCK_MAX;
//	uint32_t smem_array_max_id = smem_array_min_id + THREADBLOCK_MAX;
//
//	if (smem_array_sum_id == THREADBLOCK_MAX) {
//		printf("asdasfawdfja@opwjgf@paeo");
//
//	}
//
//	uint8_t pixel = d_domains[dblock_array_id];
//	//sum�p
//	domain_summinmax[smem_array_sum_id] = pixel;
//	//min�p
//	domain_summinmax[smem_array_min_id] = pixel;
//	//max�p
//	domain_summinmax[smem_array_max_id] = pixel;
//
//	__syncthreads();
//
//	for(int32_t i = (blockDim.x * blockDim.y) / 2; i > 0; i >>= 1){
//		if(smem_thread_id < i){
//			//sum
//			domain_summinmax[smem_array_sum_id] += domain_summinmax[smem_array_sum_id + i];
//			//min
//			if(domain_summinmax[smem_array_min_id] > domain_summinmax[smem_array_min_id + i]){
//				domain_summinmax[smem_array_min_id] = domain_summinmax[smem_array_min_id + i];
//			}
//			//max
//			if (domain_summinmax[smem_array_max_id] < domain_summinmax[smem_array_max_id + i]) {
//				domain_summinmax[smem_array_max_id] = domain_summinmax[smem_array_max_id + i];
//			}
//		}
//		__syncthreads();
//	}
//
//	//�ۑ�
//	if (dblock_thread_id == 0) {
//		dblock_sum[dblock_id] = domain_summinmax[smem_array_sum_id];
//		dblock_min[dblock_id] = domain_summinmax[smem_array_min_id];
//		dblock_max[dblock_id] = domain_summinmax[smem_array_max_id];
//	};
//}
//
///*
//	�e�����W�̑��a�C�ŏ��l�ő�l���v�Z����
//
//
//*/
//__global__ void fc_range_summimmax(uint8_t* d_ranges,
//								   uint32_t rblock_count,
//								   uint32_t* rblock_sum,
//								   uint32_t* rblock_min,
//								   uint32_t* rblock_max)
//{
//	//sum, min, max��3���ۑ�����
//	__shared__ uint32_t range_summinmax[THREADBLOCK_MAX * 3];
//	uint32_t rblock_id = blockIdx.x * blockDim.z + threadIdx.z;
//	uint32_t rblock_thread_id = threadIdx.y * blockDim.x + threadIdx.x;
//	uint32_t rblock_array_id = rblock_id + rblock_thread_id;
//
//	uint32_t smem_block_id = threadIdx.z;
//	uint32_t smem_thread_id = rblock_thread_id;
//	uint32_t smem_array_sum_id = smem_block_id * (blockDim.x * blockDim.y) + smem_thread_id;
//	uint32_t smem_array_min_id = smem_array_sum_id + THREADBLOCK_MAX;
//	uint32_t smem_array_max_id = smem_array_min_id + THREADBLOCK_MAX;
//
//	uint8_t pixel = d_ranges[rblock_array_id];
//	//sum�p
//	range_summinmax[smem_array_sum_id] = pixel;
//	//min�p
//	range_summinmax[smem_array_min_id] = pixel;
//	//max�p
//	range_summinmax[smem_array_max_id] = pixel;
//
//	__syncthreads();
//
//	for (int32_t i = (blockDim.x * blockDim.y) / 2; i > 0; i >>= 1) {
//		if (smem_thread_id < i) {
//			//sum
//			range_summinmax[smem_array_sum_id] += range_summinmax[smem_array_sum_id + i];
//			//min
//			if (range_summinmax[smem_array_min_id] > range_summinmax[smem_array_min_id + i]) {
//				range_summinmax[smem_array_min_id] = range_summinmax[smem_array_min_id + i];
//			}
//			//max
//			if (range_summinmax[smem_array_max_id] < range_summinmax[smem_array_max_id + i]) {
//				range_summinmax[smem_array_max_id] = range_summinmax[smem_array_max_id + i];
//			}
//		}
//		__syncthreads();
//	}
//
//	//�ۑ�
//	if (rblock_thread_id == 0) {
//		//if(rblock_id == 0){
//		//	printf("sum : %" PRIu32 "\n", range_summinmax[smem_array_sum_id]);
//		//	printf("min : %" PRIu32 "\n", range_summinmax[smem_array_min_id]);
//		//	printf("max : %" PRIu32 "\n", range_summinmax[smem_array_max_id]);
//		//}
//		rblock_sum[rblock_id] = range_summinmax[smem_array_sum_id];
//		rblock_min[rblock_id] = range_summinmax[smem_array_min_id];
//		rblock_max[rblock_id] = range_summinmax[smem_array_max_id];
//	};
//}
//
///*
//input:
//	uint32_t* d_dblock_sum			:�e�h���C���u���b�N�̑��a
//	uint32_t* d_dblock_min			:�e�h���C���u���b�N�̍ŏ��l
//	uint32_t* d_dblock_max			:�e�h���C���u���b�N�̍ő�l
//	uint32_t* d_rblock_sum			:�e�����W�u���b�N�̑��a
//	uint32_t* d_rblock_min			:�e�����W�u���b�N�̍ŏ��l
//	uint32_t* d_rblock_max			:�e�����W�u���b�N�̍ő�l
//	uint32_t dr_block_pixel_total	:�u���b�N���̉�f��
//output:
//	double* d_contrast_scaling		:�e�h���C���u���b�N�̊e�����W�u���b�N�ɑ΂���œK�X�P�[�����O
//	uint32_t* d_brightness_shift	:�e�h���C���u���b�N�̊e�����W�u���b�N�ɑ΂���œK�P�x�V�t�g
//
//call:
//	dim3 fc3block(THREADBLOCK_MAX);
//	dim3 fc3grid(dblock_count, rblock_count / THREADBLOCK_MAX);
//	fc_calc_scale_n_shift<<<fc3grid, fc3block>>>
//	//��̃X���b�h�u���b�N�ŕ����̃����W�u���b�N�̍ŏ��l�C�ő�l���v�Z����
//*/
//__global__ void fc_calc_scale_n_shift(uint32_t* d_dblock_sum,
//									  uint32_t* d_dblock_min,
//									  uint32_t* d_dblock_max,
//									  uint32_t* d_rblock_sum,
//									  uint32_t* d_rblock_min,
//									  uint32_t* d_rblock_max,
//									  uint32_t dr_block_pixel_total,
//									  double* d_contrast_scaling,
//									  uint32_t* d_brightness_shift)
//{
//	uint32_t dblock_id = blockIdx.x;
//	uint32_t rblock_id = blockIdx.y * blockDim.x + threadIdx.x;
//	uint32_t array_id =  blockIdx.x * (gridDim.y * blockDim.x) + rblock_id;
//
//	//�P�x�V�t�g�v�Z
//	int32_t shift = ((int32_t)d_dblock_sum[dblock_id] - (int32_t)d_rblock_sum[rblock_id])  / dr_block_pixel_total;
//	d_brightness_shift[array_id] = shift;
//	//�R���g���X�g�X�P�[�����O
//	double d = (double)(d_dblock_max[dblock_id] - d_dblock_min[dblock_id]);
//	double r = (double)(d_rblock_max[rblock_id] - d_rblock_min[rblock_id]);
//
//	double raw_scaling = r / d;
//	d_contrast_scaling[array_id] = raw_scaling;
//
//	/*
//		TODO ��{�I��4bit���ɏk������K�v���L��ׁC
//		�X�P�[�����O�̏��͈��k���ĕێ������K�v������
//	*/
//
//	//double min;
//	//double max;
//	//uint32_t scaling;
//	//for (min = -0.03125, max = 0.03125, scaling = 0; scaling < 16; min += 0.0625, max += 0.0625, scaling++) {
//	//	if (min < raw_scaling && raw_scaling < max){
//	//		d_brightness_shift[array_id] = scaling;
//	//		return;
//	//	}
//	//}
//	////0.9625�ȏ�͑S��15�ɁE�E�E�H
//	//d_brightness_shift[array_id] = 0xF;
//}
//
///*
//	dim3 fc4block(dr_blocksize, dr_blocksize, THREADBLOCK_MAX / dr_block_pixel_total);
//	dim3 fc4grid(dblock_count , rblock_count / fc4block.z);
//	fc_transform_n_calc_mse<<<fc4grid, fc4block>>>
//*/
//__global__ void fc_transform_n_calc_mse(uint8_t* d_domains,
//										uint8_t* d_ranges,
//										double* d_contrast_scaling,
//										uint32_t* d_brightness_shift,
//										uint32_t* mse)
//{
//	__shared__ uint32_t mse_all[THREADBLOCK_MAX];
//
//	uint32_t drblock_thread_id = threadIdx.y * blockDim.x + threadIdx.x;
//	uint32_t drblock_pixel_total = blockDim.x * blockDim.y;
//	uint32_t dblock_id = blockIdx.x;
//	uint32_t dblock_count = gridDim.x;
//	uint32_t rblock_id = blockIdx.y * blockDim.z + threadIdx.z;
//	uint32_t rblock_count = gridDim.y * blockDim.z;
//
//	uint32_t array_id = dblock_id * rblock_count + rblock_id;
//
//	uint32_t smem_array_id = threadIdx.z * drblock_pixel_total + drblock_thread_id;
//	uint32_t smem_block_id = threadIdx.z * drblock_pixel_total;
//	uint32_t smem_thread_id = drblock_thread_id;
//
//	//���̃����W�Ƀh���C���ɓK������scaling, shift
//	uint32_t shift = d_brightness_shift[array_id];
//	double scale = d_contrast_scaling[array_id];
//
//	double f_d_p = scale * (double)d_domains[dblock_id + drblock_thread_id] + (double)shift;
//
//	if(f_d_p < 0){
//		f_d_p = -f_d_p;
//	}
//	//uint8_t�ɃL���X�g�������_�ōő�l��255�ɌŒ肳���
//	uint8_t fixed_dpixel = (uint8_t)f_d_p;
//	int32_t fixed_dpixel2 = (int32_t)fixed_dpixel * (int32_t)fixed_dpixel;
//	//�I���W�i�������W
//	uint8_t rpixel = d_ranges[rblock_id + drblock_thread_id];
//
//	for(int32_t rotate = 0; rotate < 8; rotate++){
//		int32_t fixed_dpixel2 = (int32_t)fixed_dpixel * (int32_t)fixed_dpixel;
//		int32_t rpixel2 = (int32_t)rpixel * (int32_t)rpixel;
//		int32_t diff = fixed_dpixel2 - rpixel2;
//		diff = diff < 0 ? -diff : diff;
//		uint32_t diff_abs = diff;
//		mse_all[smem_array_id] = diff_abs;
//
//		for (int32_t j = drblock_pixel_total / 2; j > 0; j >>= 1) {
//			if(smem_thread_id < j){
//				mse_all[smem_array_id] += mse_all[smem_array_id + j];
//			}
//			__syncthreads();
//		}
//		if(smem_thread_id == 0){
//			mse[rotate * dblock_count * rblock_count + dblock_id * rblock_count + rblock_id] = mse_all[smem_array_id];
//		}
//		if (rotate < 7) {
//			if (blockDim.x == 4) {
//				rpixel = d_ranges[rblock_id + dc_affine_transform_size4[rotate][drblock_thread_id]];
//			}
//			else if (blockDim.x == 8) {
//				rpixel = d_ranges[rblock_id + dc_affine_transform_size8[rotate][drblock_thread_id]];
//			}
//			else if (blockDim.x == 16) {
//				rpixel = d_ranges[rblock_id + dc_affine_transform_size16[rotate][drblock_thread_id]];
//			}
//		}
//		__syncthreads();
//	}
//}
//
///*
//	�e�����W�̓��덷���ŏ��l�ł���h���C��(�h���܂�)��index�����_�N�V�����ŋ��߁C�e�W����ێ�����
//*/
//__global__ void fc_save_min_mse(uint32_t* d_mse, 
//								double* d_cotrast_scaling, 
//								uint32_t* d_brightness_shift, 
//								compress_data_part_rgb_gpu* d_compress_data_part_gpu) 
//{
//	
//}
//
///*
//	//
//	�摜���c��1/2�{�ɏk������
//	4�_�̕��ϒl���Z�o���邾���̕���
//	�J�[�l���Ńu���b�N�T�C�Y�����킹��
//	���u���b�N�T�C�Y��
//	���J�l�[����
//	fc_resize2<<<(grid_x, grid_y), (block_x, block_y, n)>>>(i, o, size);
//*/
//__global__ void fc_resize2(uint8_t* d_original_img, uint8_t* d_resize_img, uint32_t original_width) {
//	//blockDim.x, blockDim.y�̓��T�C�Y��̃u���b�N�T�C�Y
//	//block�̐����͕̂ϊ��O��ň��
//	uint32_t blocks_num = gridDim.x;//original_width / blockDim.x;
//
//	//resize��u���b�N�̑傫��
//	uint32_t resize_block_total = blockDim.x * blockDim.y;
//	//resize��u���b�N�̔z��̐擪index
//	uint32_t resize_block_index = (blockIdx.y * blocks_num + blockIdx.x) * resize_block_total;
//	//resize��u���b�N���X���b�h��index
//	uint32_t resize_thread_index = threadIdx.y * blockDim.y + threadIdx.x;
//
//	//���u���b�N�̑傫��
//	uint32_t orig_block_total = resize_block_total << 2;
//	//���u���b�N�̑傫���̔z��̐擪index
//	uint32_t orig_block_index = (blockIdx.y * blocks_num + blockIdx.x) * orig_block_total;
//	//resize��u���b�N���X���b�h��index1
//	uint32_t orig_thread_index1 = (threadIdx.y << 1) * (blockDim.y << 1) + (threadIdx.x << 1);
//	//resize��u���b�N���X���b�h��index2
//	uint32_t orig_thread_index2 = (threadIdx.y << 1) * (blockDim.y << 1) + (threadIdx.x << 1) + 1;
//	//resize��u���b�N���X���b�h��index3
//	uint32_t orig_thread_index3 = ((threadIdx.y << 1) + 1) * (blockDim.y << 1) + (threadIdx.x << 1);
//	//resize��u���b�N���X���b�h��index4
//	uint32_t orig_thread_index4 = ((threadIdx.y << 1) + 1) * (blockDim.y << 1) + (threadIdx.x << 1) + 1;
//
//	//resize��u���b�N���X���b�h
//	uint32_t idx1 = d_original_img[orig_block_index + orig_thread_index1];
//	uint32_t idx2 = d_original_img[orig_block_index + orig_thread_index2];
//	uint32_t idx3 = d_original_img[orig_block_index + orig_thread_index3];
//	uint32_t idx4 = d_original_img[orig_block_index + orig_thread_index4];
//
//	d_resize_img[resize_block_index + resize_thread_index] = (uint8_t)((idx1 + idx2 + idx3 + idx4) >> 2);
//}
//
///*
//	�S�̂�4x4�C�u���b�N�T�C�Y��2x2�̎�
//	{ 0, 1, 2, 3,
//	  4, 5, 6, 6,
//	  8, 9,10,11,
//	 12,13,14,14}
//	�̂悤�ȉ�f�l�̕��т̃O���[�X�P�[���̔z��摜���e�X�g�\������
//	�S��f�������Ă����̂ő��x�͒x���C�e�X�g�p
//*/
//void show_img(uint8_t* img_array, uint32_t width, uint32_t height){
//	cv::Mat mat(width, height, CV_8U);
//	for (uint32_t y = 0; y < height; y++) {
//		for (uint32_t x = 0; x < width; x++) {
//			mat.at<uint8_t>(y, x) = img_array[y*width + x];
//		}
//	}
//	cv::namedWindow("show_img", cv::WINDOW_AUTOSIZE);
//	cv::imshow("show_img", mat);
//	cv::waitKey(0);
//	cv::destroyAllWindows();
//}
//
///*
//	�S�̂�4x4�C�u���b�N�T�C�Y��2x2�̎�
//	{ 0, 1, 4, 5,
//	  2, 3, 6, 7,
//	  8, 9,12,13,
//	 10,11,14,15}
//	�̂悤�ȉ�f�l�̕��т̃O���[�X�P�[���̔z��摜���e�X�g�\������
//	�S��f�������Ă����̂ő��x�͒x���C�e�X�g�p
//	block_size == block_height == block_width
//*/
//void show_img2(uint8_t* img_array, uint32_t width, uint32_t height, uint32_t block_size) {
//	cv::Mat mat(width, height, CV_8U);
//
//	//�������̃����W�̐�
//	uint32_t range_x_n = width / block_size;
//	//�c�����̃����W�̐�
//	uint32_t range_y_n = height / block_size;
//
//	uint32_t index = 0;
//	for(uint32_t y = 0; y < range_y_n * block_size; y += block_size){
//		for (uint32_t x = 0; x < range_x_n * block_size; x += block_size) {
//			//�e�u���b�N
//			for (uint32_t block_rows = 0; block_rows < block_size; block_rows++) {
//				for (uint32_t block_cols = 0; block_cols < block_size; block_cols++) {
//					mat.at<uint8_t>(y + block_rows, x + block_cols) = img_array[index];
//					index++;
//				}
//			}
//		}
//	}
//
//	cv::namedWindow("show_img", cv::WINDOW_AUTOSIZE);
//	cv::imshow("show_img", mat);
//	cv::waitKey(0);
//	cv::destroyAllWindows();
//}
//
//
///*
//	CPU��
//	�S�̂�4x4�C�u���b�N�T�C�Y��2x2�̎��C��f�l�̕��т�1���ȉ��̔z��̕��тɂ���
//	{ 0, 1, 2, 3,
//	  4, 5, 6, 6,
//	  8, 9,10,11,
//	 12,13,14,14} -> {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15}
//*/
//void img2array(cv::Mat img, uint8_t* img_array) {
//	assert(img.isContinuous());
//	img.convertTo(img, CV_8UC1);
//
//
//	for (uint32_t y = 0; y < img.rows; y++) {
//		for (uint32_t x = 0; x < img.cols; x++) {
//			img_array[y*img.rows + x] = img.at<uint8_t>(y, x);
//		}
//	}
//}
//
///*
//	CPU��
//	�S�̂�4x4�C�u���b�N�T�C�Y��2x2�̎��C��f�l�̕��т�1���ȉ��̔z��̕��тɂ���
//	{ 0, 1, 2, 3,
//	  4, 5, 6, 7,
//	  8, 9,10,11,
//	 12,13,14,15} -> {0, 1, 4, 5, 2, 3, 6, 7, 8, 9, 12, 13, 10, 11, 14, 15}
//*/
//void img2blockarray(cv::Mat img, uint8_t* img_array, uint32_t block_size){
//	assert(img.isContinuous());
//	assert((img.cols % block_size) == 0);
//	assert((img.rows % block_size) == 0);
//	img.convertTo(img, CV_8UC1);
//
//	//�������̃����W�̐�
//	uint32_t range_x_n = img.cols / block_size;
//	//�c�����̃����W�̐�
//	uint32_t range_y_n = img.rows / block_size;
//	std::cout << "tatal : "<<img.total() << std::endl;
//	uint32_t index = 0;
//	for (uint32_t y = 0; y < range_y_n * block_size; y += block_size) {
//		for (uint32_t x = 0; x < range_x_n * block_size; x += block_size) {
//			//�e�u���b�N
//			
//			for (uint32_t block_rows = 0; block_rows < block_size; block_rows++) {
//				for (uint32_t block_cols = 0; block_cols < block_size; block_cols++) {
//					
//					//if(index % 1000 == 0)std::cout << index << std::endl;
//					img_array[index] = img.at<uint8_t>(y + block_rows, x + block_cols);
//					index++;
//				}
//			}
//		}
//	}
//
//}
//
//void launch_rgb_compress_kernel(cv::Mat img, uint32_t blocksize)
//{	
//	/*
//		0.�O�����
//	*/
//
//	//�h���C���E�����W�u���b�N�̂P�ӂ̒���
//	uint32_t dr_blocksize = blocksize;
//	//�u���b�N1���܂މ�f��
//	uint32_t dr_block_pixel_total = dr_blocksize * dr_blocksize;
//	//�����W�u���b�N�̕ӓ�����̐�
//	uint32_t rblock_cols = img.cols / dr_blocksize;
//	uint32_t rblock_rows = img.rows / dr_blocksize;
//	uint32_t rblock_count = rblock_cols * rblock_rows;
//	//�h���C���u���b�N�̕ӕӂ�̐�
//	uint32_t dblock_cols = rblock_cols >> 1;
//	uint32_t dblock_rows = rblock_rows >> 1;
//	uint32_t dblock_count = dblock_cols * dblock_rows;
//
//	assert(blocksize == 4 || blocksize == 8 || blocksize == 16);
//	assert((img.cols % blocksize) == 0);
//	assert((img.rows % blocksize) == 0);
//	assert(dblock_count % (THREADBLOCK_MAX / blocksize) == 0);
//	assert(rblock_count % (THREADBLOCK_MAX / blocksize) == 0);
//	assert(((rblock_count * dblock_count) % THREADBLOCK_MAX) == 0);
//	assert(rblock_count < (65535 * THREADBLOCK_MAX));
//
//	std::cout << "domain block count : " << dblock_count << std::endl;
//	std::cout << "range block count : " << rblock_count << std::endl;
//
//	/*
//		1.�u���b�N�ϊ��E�k���ϊ�
//	*/
//	//�u���b�N�ϊ��E�k���ϊ��ׂ̈̋N���X���b�h���ݒ�
//	dim3 fc1block(dr_blocksize, dr_blocksize);
//	dim3 fc1grid(rblock_cols, rblock_rows);
//	std::cout << "reduction grid :" << " x = " << fc1grid.x << " y = " << fc1grid.y << std::endl;
//	uint32_t orig_arraysize = img.total() * img.channels();
//	uint8_t* h_orig_img = new uint8_t[orig_arraysize];
//	uint8_t* d_orig_img;
//	uint8_t* d_ranges;
//	uint8_t* d_domains;
//	
//	CHECK(cudaMalloc((void**)&d_orig_img, sizeof(uint8_t) * orig_arraysize));
//	CHECK(cudaMalloc((void**)&d_ranges, sizeof(uint8_t) * orig_arraysize));
//	CHECK(cudaMalloc((void**)&d_domains, sizeof(uint8_t) * orig_arraysize >> 2));
//
//	img2array(img, h_orig_img);
//
//	CHECK(cudaMemcpy(d_orig_img, h_orig_img, sizeof(uint8_t) * orig_arraysize, cudaMemcpyHostToDevice));
//	fc_make_domains_n_ranges<<<fc1grid, fc1block>>>(d_orig_img, d_ranges, d_domains);
//	CHECK(cudaDeviceSynchronize());
//	//uint8_t* h_ranges = new uint8_t[orig_arraysize];
//	//uint8_t* h_domains = new uint8_t[orig_arraysize >> 2];
//	//CHECK(cudaMemcpy(h_ranges, d_ranges, sizeof(uint8_t) * orig_arraysize, cudaMemcpyDeviceToHost));
//	//CHECK(cudaMemcpy(h_domains, d_domains, sizeof(uint8_t) * orig_arraysize >> 2, cudaMemcpyDeviceToHost));
//	//show_img2(h_ranges, img.cols, img.rows, dr_blocksize);
//	//show_img2(h_domains, img.cols >> 1, img.rows >> 1, dr_blocksize);
//
//	/*
//		2.�h���C���E�����W�̑��a�E�ŏ��l�ő�l�v�Z
//	*/
//	//�P�̃X���b�h�u���b�N�ŕ����̃h���C������������
//	dim3 fc2dblock(dr_blocksize, dr_blocksize, THREADBLOCK_MAX / dr_block_pixel_total);
//	dim3 fc2dgrid(dblock_count / fc2dblock.z);
//
//	uint32_t* d_dblock_sum;
//	uint32_t* d_dblock_min;
//	uint32_t* d_dblock_max;
//	CHECK(cudaMalloc((void**)&d_dblock_sum, sizeof(uint32_t) * dblock_count));
//	CHECK(cudaMalloc((void**)&d_dblock_min, sizeof(uint32_t) * dblock_count));
//	CHECK(cudaMalloc((void**)&d_dblock_max, sizeof(uint32_t) * dblock_count));
//	fc_domain_summimmax<<<fc2dgrid, fc2dblock>>>(d_domains, dblock_count, d_dblock_sum, d_dblock_min, d_dblock_max);
//
//	//�P�̃X���b�h�u���b�N�ŕ����̃����W����������
//	dim3 fc2rblock(dr_blocksize, dr_blocksize, THREADBLOCK_MAX / dr_block_pixel_total);
//	dim3 fc2rgrid(rblock_count / fc2rblock.z);
//	uint32_t* d_rblock_sum;
//	uint32_t* d_rblock_min;
//	uint32_t* d_rblock_max;
//	CHECK(cudaMalloc((void**)&d_rblock_sum, sizeof(uint32_t) * rblock_count));
//	CHECK(cudaMalloc((void**)&d_rblock_min, sizeof(uint32_t) * rblock_count));
//	CHECK(cudaMalloc((void**)&d_rblock_max, sizeof(uint32_t) * rblock_count));
//	fc_range_summimmax<<<fc2rgrid, fc2rblock>>>(d_ranges, rblock_count, d_rblock_sum, d_rblock_min, d_rblock_max);
//
//	CHECK(cudaDeviceSynchronize());
//
//	/*
//		3.�R���g���X�g�X�P�[�����O�E�P�x�V�t�g�v�Z
//	*/
//
//	dim3 fc3block(THREADBLOCK_MAX);
//	dim3 fc3grid(dblock_count, rblock_count / THREADBLOCK_MAX);
//
//	double* d_contrast_scaling;
//	uint32_t* d_brightness_shift;
//
//	CHECK(cudaMalloc((void**)&d_contrast_scaling, sizeof(double) * dblock_count * rblock_count));
//	CHECK(cudaMalloc((void**)&d_brightness_shift, sizeof(uint32_t) * dblock_count * rblock_count));
//	//CHECK(cudaMalloc((void**)&d_adjust_domains_for_ranges, sizeof(uint32_t) * dblock_count * rblock_count * dr_block_pixel_total));
//
//	fc_calc_scale_n_shift<<<fc3grid, fc3block>>>(d_dblock_sum,
//												 d_dblock_min,
//												 d_dblock_max,
//												 d_rblock_sum,
//									     		 d_rblock_min,
//												 d_rblock_max,
//												 dr_block_pixel_total,
//											     d_contrast_scaling,
//											     d_brightness_shift);
//
//	CHECK(cudaDeviceSynchronize());
//
//	/*
//		4.�R���g���X�g�X�P�[�����O�E�P�x�V�t�g�K�p�E��]�E�����ϊ��E���v�Z�EMSE�v�Z�i���_�N�V�����j
//	*/
//
//	//
//	dim3 fc4block(dr_blocksize, dr_blocksize, THREADBLOCK_MAX / dr_block_pixel_total);
//	dim3 fc4grid(dblock_count , rblock_count / fc4block.z);
//	//�e�����W�̊e�h���C���̊e��]�ϊ����MSE��ۑ����Ă���
//	uint32_t* d_mse;
//	CHECK(cudaMalloc((void**)&d_mse, sizeof(uint32_t) * rblock_count * dblock_count * 8));
//
//	fc_transform_n_calc_mse<<<fc4grid, fc4block>>>(d_domains,
//												   d_ranges,
//												   d_contrast_scaling,
//												   d_brightness_shift,
//												   d_mse);
//	CHECK(cudaDeviceSynchronize());
//	//std::cout << sizeof(uint32_t) * rblock_count * dblock_count * 8 << "byte" << std::endl;
//
//	//uint32_t* h_mse = new uint32_t[rblock_count * dblock_count * 8];
//	//CHECK(cudaMemcpy(h_mse, d_mse, sizeof(uint32_t) * rblock_count * dblock_count * 8, cudaMemcpyDeviceToHost));
//
//	//std::cout << "nukiuti : " << h_mse[231] << std::endl;
//
//	//delete[] h_mse;
//
//	/*
//		5.�e�����W���ŏ�MSE�Eindex�v�Z(���_�N�V����)
//	*/
//	//�u���b�N�T�C�Y�ɉ������R���X�^���g������������������
//	init_affine_transformer(dr_blocksize);
//	dim3 fc5block(THREADBLOCK_MAX);
//	dim3 fc5grid(dblock_count * 8, rblock_count/ THREADBLOCK_MAX);
//
//	//�����W�̐��������k�f�[�^��ۑ�����
//	compress_data_part_rgb_gpu* h_compress_data = new compress_data_part_rgb_gpu[rblock_count];
//	compress_data_part_rgb_gpu* d_compress_data;
//	CHECK(cudaMalloc((void**)&d_compress_data, sizeof(compress_data_part_rgb_gpu) * rblock_count));
//
//	fc_save_min_mse<<<fc5grid, fc5block>>>(d_mse, d_contrast_scaling, d_brightness_shift, d_compress_data);
//	
//	CHECK(cudaMemcpy(h_compress_data, d_compress_data, sizeof(compress_data_part_rgb_gpu) * rblock_count, cudaMemcpyDeviceToHost));
//
//	//delete[] h_ranges;
//	//delete[] h_domains;
//
//	/*
//		L.�㏈��
//	*/
//
//	delete[] h_orig_img;
//	delete[] h_compress_data;
//
//	CHECK(cudaFree(d_orig_img));
//	CHECK(cudaFree(d_ranges));
//	CHECK(cudaFree(d_domains));
//
//	CHECK(cudaFree(d_rblock_sum));
//	CHECK(cudaFree(d_rblock_min));
//	CHECK(cudaFree(d_rblock_max));
//	
//	CHECK(cudaFree(d_dblock_sum));
//	CHECK(cudaFree(d_dblock_min));
//	CHECK(cudaFree(d_dblock_max));
//
//	CHECK(cudaFree(d_contrast_scaling));
//	CHECK(cudaFree(d_brightness_shift));
//
//	CHECK(cudaFree(d_mse));
//
//	CHECK(cudaDeviceSynchronize());
//}
