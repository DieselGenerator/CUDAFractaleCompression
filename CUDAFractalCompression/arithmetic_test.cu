#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <inttypes.h>

#include <hip/hip_runtime.h>
#include <>

#include "arithmetic_test.cuh"

#include "cuda_call_checker.cuh"

__global__ void abs1(int* d_forabs){

	int x_tid = threadIdx.x;
	int y_tid = threadIdx.y;
	int my_abs = x_tid - y_tid;

	volatile int i = my_abs < 0 ? -my_abs : my_abs;
}


__global__ void abs2(int* d_forabs) {

	int x_tid = threadIdx.x;
	int y_tid = threadIdx.y;
	int my_abs = x_tid - y_tid;

	volatile int i = (my_abs ^ (my_abs >> 31)) - (my_abs >> 31);

}

__global__ void max() {
	uint8_t ui8 = 0;
	double d = 1000;
	ui8 = (uint8_t)d;
	printf("uint8_t value : %d \n", ui8);
}

void arithmetic_speedtest(){
	
	CHECK(hipDeviceReset());

	std::cout << "size ui" <<  sizeof(unsigned int) << std::endl;
	std::chrono::system_clock::time_point start, end;
	int forabs[64];
	int *d_forabs;
	//33554432,64,64
	dim3 grid(10000, 64, 64);
	dim3 block(8, 8);
	
	//dim3 grid(1);
	//dim3 block(32, 32, 2);
	CHECK(hipMalloc((void**)&d_forabs, sizeof(int) * 64));

	start = std::chrono::system_clock::now();

	abs1<<<grid, block>>>(d_forabs);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf("Error: %s\n", hipGetErrorString(err));

	CHECK(hipDeviceSynchronize());

	end = std::chrono::system_clock::now();
	auto elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
	std::cout << "abs1 elapsed " << elapsed << " milli sec \n";

	CHECK(hipMemcpy(forabs, d_forabs, sizeof(int) * 64, hipMemcpyDeviceToHost));
	CHECK(hipDeviceSynchronize());

	//abs2

	start = std::chrono::system_clock::now();

	abs2<<<grid, block>>>(d_forabs);
	err = hipGetLastError();
	if (err != hipSuccess)
		printf("Error: %s\n", hipGetErrorString(err));

	CHECK(hipDeviceSynchronize());

	end = std::chrono::system_clock::now();
	elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
	std::cout << "abs2 elapsed " << elapsed << " milli sec \n";

	CHECK(hipMemcpy(forabs, d_forabs, sizeof(int) * 64, hipMemcpyDeviceToHost));
	CHECK(hipDeviceSynchronize());

	//max
	max<<<1, 1>>>();
	CHECK(hipDeviceSynchronize());
	



	//for(int i = 0; i < 64; i++){
	//	std::cout << forabs[i] << std::endl;
	//}
}
